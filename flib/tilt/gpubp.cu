#include "hip/hip_runtime.h"
/*
 *  gpubp.cu -- Kernel and C code for CUDA-based backprojection, reprojection
 *               and Fourier filtering
 *
 *  Author: David Mastronarde   email: mast@colorado.edu
 *
 *  Copyright (C) 2009 by Boulder Laboratory for 3-Dimensional Electron
 *  Microscopy of Cells ("BL3DEMC") and the Regents of the University of 
 *  Colorado.  See dist/COPYRIGHT for full copyright notice.
 *
 *  $Id$
 *  Log at end of file
 */
#include <stdio.h>
#include <stdarg.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "imodconfig.h"

#ifdef F77FUNCAP
#define gpuavailable GPUAVAILABLE
#define gpuallocarrays GPUALLOCARRAYS
#define gpuloadproj GPULOADPROJ
#define gpushiftproj GPUSHIFTPROJ
#define gpubpxtilt GPUBPXTILT
#define gpubpnox GPUBPNOX
#define gpubplocal GPUBPLOCAL
#define gpuloadlocals GPULOADLOCALS
#define gpuloadfilter GPULOADFILTER
#define gpufilterlines GPUFILTERLINES
#define gpureproject GPUREPROJECT
#define gpureprojlocal GPUREPROJLOCAL
#define gpureprojoneslice GPUREPROJONESLICE
#define gpudone GPUDONE
#else
#define gpuavailable gpuavailable_
#define gpuallocarrays gpuallocarrays_
#define gpuloadproj gpuloadproj_
#define gpushiftproj gpushiftproj_
#define gpubpxtilt gpubpxtilt_
#define gpubpnox gpubpnox_
#define gpubplocal gpubplocal_
#define gpuloadlocals gpuloadlocals_
#define gpuloadfilter gpuloadfilter_
#define gpufilterlines gpufilterlines_
#define gpureproject gpureproject_
#define gpureprojlocal gpureprojlocal_
#define gpureprojoneslice gpureprojoneslice_
#define gpudone gpudone_
#endif

#ifdef __cplusplus
extern "C" {
  int gpuavailable(int *nGPU, float *memory, int *debug);
  int gpuallocarrays(int *width, int *nyout, int *nxprj2, int *nyprj,
                     int *nplanes, int *nviews, int *numWarps, int *numDelz,
                     int *nfilt, int *nreproj, int *firstNpl, int *lastNpl);
  int gpubpnox(float *slice, float *lines, float *sbeta, float *cbeta,
               int *nxprj,
               float *xcenin, float *xcen, float *ycen, float *edgefill);
  int gpushiftproj(int *numPlanes, int *lsliceStart, int *loadStart);
  int gpuloadproj(float *lines, int *numPlanes, int *lsliceStart, 
                  int *loadStart);
  int gpubpxtilt(float *slice, float *sbeta, float *cbeta, float *salpha,
                 float *calpha, float *xzfac, float *yzfac, int *nxprj,
                 int *nyprj, float *xcenin, float *xcen, float *ycen,
                 int *lslice, float *slicen, float *edgefill);
  int gpubplocal(float *slice, int *lslice, int *nxwarp, int *nywarp,
                 int *ixswarp, int *iyswarp, int *idxwarp, int *idywarp,
                 int *nxprj, float *xcen, float *xcenin, float *delxx,
                 float *ycen, float *slicen, float *edgefill);
  int gpuloadfilter(float *lines);
  int gpuloadlocals(float *packed, int *numWarps);
  int gpufilterlines(float *lines, int *lslice, int *filterSet);
  int gpureproject(float *lines, float *sbeta, float *cbeta, float *salpha, 
                   float *calpha, float *xzfac, float *yzfac, float *delz,
                   int *lsStart, int *lsEnd, int *ithick,
                   float *xcen, float *xcenPdelxx, int *minXreproj, 
                   float *xprjOffset, float *ycen, int *minYreproj,
                   float *yprjOffset, float *slicen, int *ifalpha, 
                   float *pmean);
  int gpureprojoneslice(float *slice, float *lines, float *sbeta, float *cbeta,
                        float *ycen, int *numproj, float *pmean);
  int gpureprojlocal
  (float *lines, float *sbeta, float *cbeta, float *salpha, float *calpha,
   float *xzfac, float *yzfac, int *nxwarp, int *nywarp, int *ixswarp, 
   int *iyswarp, int *idxwarp, int *idywarp, float *warpDelz, int *nWarpDelz, 
   float *dxWarpDelz,float *xprojMin,float *xprojMax, int *lsStart, int *lsEnd,
   int *ithick, int *iview, float *xcen, float *xcenin, float *delxx, 
   int *minXload, float *xprjOffset, float *ycenAdj, float *yprjOffset,
   float *slicen, float *pmean);
  void gpudone();
}
#endif

static int checkProjLoad(int *numPlanes, int *lsliceStart, int startm1);
static int testReportErr(char *mess);
static int loadBetaInvertCos(float *cbeta, float *sbeta, float *costmp,
                             int num);
static int synchronizeCopySlice(float *devslc, int pitch, float *slice,
                                int width, int numLines);
static void pflush(const char *format, ...);
static void pflerr(const char *format, ...);
static void allocerr(char *mess, int *nplanes, int *firstNpl,
                     int *lastNpl, int ifcuda);



// Offsets to positions in constant array
// For some reason 6 separate arrays did not work for xtilt case
#define DELTA_OFS  720
#define MAX_TABLE (6 * DELTA_OFS)
__constant__ float tables[MAX_TABLE];
__constant__ int rpNumz[DELTA_OFS];

#define COSOFS 0
#define SINOFS (1 * DELTA_OFS)
#define CALOFS (2 * DELTA_OFS)
#define SALOFS (3 * DELTA_OFS)
#define XZFOFS (4 * DELTA_OFS)
#define YZFOFS (5 * DELTA_OFS)
#define INVOFS (2 * DELTA_OFS)

// Definitions for accessing the local alignments arrays with texture calls
#define F11IND 0.f
#define F21IND 1.f
#define F12IND 2.f
#define F22IND 3.f
#define F13IND 4.f
#define F23IND 5.f
#define CAIND 6.f
#define SAIND 7.f
#define CBIND 8.f
#define SBIND 9.f
#define XZFIND 10.f
#define YZFIND 11.f


// declare texture reference for 2D float textures
texture<float, 2, hipReadModeElementType> projtex;
texture<float, 2, hipReadModeElementType> localtex;
texture<float, 2, hipReadModeElementType> rpSlicetex;
texture<float, 2, hipReadModeElementType> pfactex;
texture<float, 2, hipReadModeElementType> delztex;

// Static variables for device arrays
static float *devSlice = NULL;
static hipArray* devProj = NULL;
static float *xprojf = NULL;
static float *xprojz = NULL;
static float *yprojf = NULL;
static float *yprojz = NULL;
static hipArray *localData = NULL;
static hipArray *localPfac = NULL;
static hipArray *devDelz = NULL;
static float *radialFilt = NULL;
static float *devFFT = NULL;
static hipArray *devRpSlice = NULL;
static float *devReproj = NULL;

// Other static variables
static hipfftHandle forwardPlan = 0, inversePlan = 0;
static int max_gflops_device = -1;
static int deviceSelected = 0;
static size_t slicePitch;
static size_t reprojPitch;
static size_t localPitch;
static int sliceThick, sliceWidth, numViews, numProjPlanes;
static int lsliceFirst, numLoadedPlanes, nxPlane, nyPlane, numFilts;
static int copyFilteredOK = 0;
static int *planeLoaded;

/*
 *  SETUP/SHUTDOWN ROUTINES
 */

/*
 * Test whether a GPU is available, either a GPU of the given number if nGPU is
 * > 0, or the one with the best processing rate if nGPU is 0, and return the
 * memory in bytes.  Return value is 1 for success, 0 for failure.
 */
int gpuavailable(int *nGPU, float *memory, int *debug)
{
  int current_device = 0;
  int device_count = 0;
  float gflops;
  struct hipDeviceProp_t device_properties, best_properties;
  float max_gflops = 0;
  *memory = 0;
  hipGetDeviceCount( &device_count );
  if (*debug)
    pflush("Device count = %d\n", device_count);
  if (*nGPU != 0) {
    if (*nGPU < 0 || *nGPU > device_count)
      return 0;
    current_device = *nGPU - 1;
    device_count = *nGPU;
  }
  for (; current_device < device_count; current_device++) {
    if (hipGetDeviceProperties( &device_properties, current_device)
        != hipSuccess) {
      pflerr("Error returned from trying to get properties of GPU device %d",
               current_device);
      return 0;
    }
    if (*debug)
      pflush("Device %d: mpc %d  cr %d  major %d minor %d  mem %.0f\n",
             current_device, device_properties.multiProcessorCount,
             device_properties.clockRate, device_properties.major,
             device_properties.minor, (float)device_properties.totalGlobalMem);
    gflops = device_properties.multiProcessorCount * 
      device_properties.clockRate;

    // Exclude emulation mode (?) which shows up on the Mac
    if( gflops > max_gflops && device_properties.major != 9999) {
      max_gflops = gflops;
      max_gflops_device = current_device;
      best_properties = device_properties;
    }
  }
    
  if (max_gflops_device >= 0) {
    *memory = best_properties.totalGlobalMem;
    return 1;
  }
  return 0;
}

/*
 * Allocate all needed arrays on the GPU.  Allocate a reconstructed slice or
 * reprojected line array of width x nyout, an array for nplanes of input data 
 * each with nyprj lines of length nxprj2, and local alignment arrays if
 * numWarps > 0.  If numDelz > 0, this indicates reprojection of with local
 * alignments and causes local projection factor arrays to be allocated for
 * nplanes lines, allocation of a CUDA array for those factors too, and 
 * allocation of an array of numDelz x nplanes for warpDelz values.  If 
 * nfilt > 0, also allocate arrays for line filtering with nfilt sets of
 * filters.  If nreproj > 0, allocate separate arrays for reprojecting one
 * slice while still doing regular backprojection arrays.
 */
int gpuallocarrays(int *width, int *nyout, int *nxprj2, int *nyprj,
                   int *nplanes, int *nviews, int *numWarps, int *numDelz,
                   int *nfilt, int *nreproj, int *firstNpl, int *lastNpl)
{
  size_t pitch1, pitch2, pitch3, memTot;
  int nlines;

  if (max_gflops_device < 0)
    return 1;
  if (!deviceSelected && hipSetDevice(max_gflops_device) != hipSuccess) {
    allocerr("Error selecting GPU device", nplanes, firstNpl, lastNpl, 1);
    return 1;
  }
  deviceSelected = 1;

  // Allocate memory for slice or reprojected lines on device
  size_t sizetmp = *width * sizeof(float);
  if (hipMallocPitch((void **)&devSlice, &slicePitch, sizetmp, *nyout) != 
      hipSuccess) {
    allocerr("Failed to allocate slice array on GPU device", nplanes, 
             firstNpl, lastNpl, 1);
    return 1;
  }
  //pflush("reproj array size %d %d\n", *width, *nyout);

  // Allocate memory for projection lines or slices to reproject
  hipChannelFormatDesc projDesc = hipCreateChannelDesc
    (32, 0, 0, 0, hipChannelFormatKindFloat);
  if (hipMallocArray(&devProj, &projDesc, *nxprj2, *nyprj * *nplanes)
      != hipSuccess) {
    pflush("malloc %d %d %d %d\n", *nxprj2, *nyprj, *nplanes,
           *nyprj * *nplanes);
    allocerr("Failed to allocate projection array on GPU device", nplanes, 
             firstNpl, lastNpl, 1);
    return 1;
  }
  memTot = sizetmp * *nyout + 4 * *nxprj2 * *nyprj * *nplanes;
  //pflush("input slice array size %d %d %d\n", *nxprj2, *nyprj, *nplanes);

  // set texture parameters
  projtex.addressMode[0] = hipAddressModeClamp;
  projtex.addressMode[1] = hipAddressModeClamp;
  projtex.filterMode = hipFilterModeLinear;
  projtex.normalized = false;
  
  // Bind the array to the texture
  if (hipBindTextureToArray(projtex, devProj, projDesc) != hipSuccess) {
    allocerr("Failed to bind projection array to texture", nplanes, firstNpl,
             lastNpl, 1);
    return 1;
  }

  if (*nplanes > 1) {
    planeLoaded = (int *)malloc(*nplanes * sizeof(int));
    if (!planeLoaded) {
      allocerr("Failed to malloc little array planeLoaded\n", nplanes,
               firstNpl, lastNpl, 0);
      return 1;
    }
  }

  // Get arrays for reprojection of one slice
  if (*nreproj) {
    if (hipMallocArray(&devRpSlice, &projDesc, *width, *nyout) !=
        hipSuccess) {
      allocerr("Failed to allocate slice array for reprojection on GPU device",
               nplanes, firstNpl, lastNpl, 1);
      return 1;
    }
    if (hipBindTextureToArray(rpSlicetex,devRpSlice,projDesc) != hipSuccess){
      allocerr("Failed to bind reprojection slice array to texture", nplanes, 
               firstNpl, lastNpl, 1);
      return 1;
    }
    rpSlicetex.addressMode[0] = hipAddressModeClamp;
    rpSlicetex.addressMode[1] = hipAddressModeClamp;
    rpSlicetex.filterMode = hipFilterModeLinear;
    rpSlicetex.normalized = false;
  
    if (hipMallocPitch((void **)&devReproj, &reprojPitch, 
                        (size_t)(*nxprj2 * sizeof(float)), *nreproj)
        != hipSuccess) {
      allocerr("Failed to allocate reprojected line array on GPU device", 
               nplanes, firstNpl, lastNpl, 1);
      return 1;
    }
    memTot += 4 * *width * *nyout + *nxprj2 * *nreproj;
  }

  // Get arrays for local proj factors
  if (*numWarps > 0) {
    nlines = *nyprj;

    // Adjust and allocate for reprojection
    if (*numDelz) {
      nlines = *nplanes;
      sizetmp = *nxprj2 * sizeof(float);

      if (hipMallocArray(&localPfac, &projDesc, *nxprj2, 4 * nlines) !=
          hipSuccess) {
        allocerr("Failed to allocate local factor texture array on GPU device"
                 , nplanes, firstNpl, lastNpl, 1);
        return 1;
      }
      //pflush("local factor texture  %d %d\n", *nxprj2, 4 * nlines);
      
      pfactex.filterMode = hipFilterModePoint;
      pfactex.normalized = false;
      if (hipBindTextureToArray(pfactex, localPfac, projDesc) != hipSuccess){
        allocerr("Failed to bind local factor arrays to texture", nplanes, 
                 firstNpl, lastNpl, 1);
        return 1;
      }
      if (hipMallocArray(&devDelz, &projDesc, *numDelz, nlines) != 
          hipSuccess) {
        allocerr("Failed to allocate warpDelz texture array on GPU device",
                 nplanes, firstNpl, lastNpl, 1);
        return 1;
      }
      //pflush("warpdelz texture  %d %d\n", *numDelz, nlines);
      delztex.filterMode = hipFilterModePoint;
      delztex.normalized = false;
      if (hipBindTextureToArray(delztex, devDelz, projDesc) != hipSuccess) {
        allocerr("Failed to bind warpDelz array to texture", nplanes, 
                 firstNpl, lastNpl, 1);
        return 1;
      }
      memTot += 4 * nlines * (4 * *nxprj2 + *numDelz);
    }

    // Allocate the arrays always used for local data
    if (hipMallocPitch((void **)&xprojf, &pitch1, sizetmp, nlines) != 
        hipSuccess ||
        hipMallocPitch((void **)&xprojz, &pitch2, sizetmp, nlines) != 
        hipSuccess ||
        hipMallocPitch((void **)&yprojf, &pitch3, sizetmp, nlines) != 
        hipSuccess ||
        hipMallocPitch((void **)&yprojz, &localPitch, sizetmp, nlines) != 
        hipSuccess  ||
        hipMallocArray(&localData, &projDesc, *numWarps * *nviews, 12) 
        != hipSuccess) {
      allocerr("Failed to allocate local factor arrays on GPU device", nplanes,
               firstNpl, lastNpl, 1);
      return 1;
    }
    /* pflush("xyprojf pitches  %d %d    localdata %d\n", *nxprj2, nlines,
     *numWarps * *nviews); */
    if (pitch2 != pitch1 || pitch3 != pitch1 || localPitch != pitch1) {
      allocerr("Array pitches for local GPU arrays do NOT match\n", nplanes,
               firstNpl, lastNpl, 0);
      return 1;
    }

    localtex.filterMode = hipFilterModePoint;
    localtex.normalized = false;
    if (hipBindTextureToArray(localtex, localData, projDesc) != hipSuccess) {
      allocerr("Failed to bind local factor arrays to texture", nplanes, 
               firstNpl, lastNpl, 1);
      return 1;
    }
    memTot += 4 * sizetmp * nlines + 48 * *numWarps * *nviews;
  }

  // Get arrays for radial filtering
  if (*nfilt > 0) {
    sizetmp = *nxprj2 * *nyprj * sizeof(float);
    if (hipMalloc((void **)&devFFT, sizetmp)  != hipSuccess ||
        hipMalloc((void **)&radialFilt, sizetmp * *nfilt)  != hipSuccess) {
      allocerr("Failed to allocate GPU arrays for radial filtering", nplanes,
               firstNpl, lastNpl, 1);
      return 1;
    }
    memTot += (1 + *nfilt) * sizetmp;
    numFilts = *nfilt;
  }

  pflush("Allocated %4d MB for arrays (including %d input planes) on the GPU\n"
         , (memTot + 512*1024)/(1024*1024), *nplanes);
  sliceWidth = *width;
  sliceThick = *nyout;    // Only good for backprojection!
  numViews = *nviews;
  numProjPlanes = *nplanes;
  nxPlane = *nxprj2;
  nyPlane = *nyprj;
  return 0;
}

// Routine to free all allocated resources
void gpudone()
{
  hipFree(devSlice);
  hipFreeArray(devProj);
  hipFree(xprojf);
  hipFree(xprojz);
  hipFree(yprojf);
  hipFree(yprojz);
  hipFreeArray(localData);
  hipFreeArray(localPfac);
  hipFreeArray(devDelz);
  hipFree(devFFT);
  hipFree(radialFilt);
  hipFree(devReproj);
  hipFreeArray(devRpSlice);
  if (forwardPlan)
    hipfftDestroy(forwardPlan);
  if (inversePlan)
    hipfftDestroy(inversePlan);
  devSlice = NULL;
  devProj = NULL;
  xprojf = NULL;
  xprojz = NULL;
  yprojf = NULL;
  yprojz = NULL;
  localData = NULL;
  localPfac = NULL;
  devDelz = NULL;
  devFFT = NULL;
  radialFilt = NULL;
  devReproj = NULL;
  devRpSlice = NULL;
  forwardPlan = 0;
  inversePlan = 0;
}

/*
 * ROUTINES FOR LOADING/MAINTAINING STACK OF PLANES ON GPU
 */ 

// Function to shift existing data in preparation for loading new data starting
// in position loadStart (numbered from 1) and with starting slice number
// lsliceStart
int gpushiftproj(int *numPlanes, int *lsliceStart, int *loadStart)
{
  int startm1 = *loadStart - 1;
  int shift, shiftStart, numToShift, todo, dstY, srcY;
  size_t sizetmp = nxPlane * sizeof(float);
  if (startm1 > 0) {
    if (checkProjLoad(numPlanes, lsliceStart, startm1))
      return 1;

    // Copy data down without overlap if it goes into occupied planes
    if (startm1 < numLoadedPlanes) {
      shift = numLoadedPlanes - startm1;
      numToShift = startm1;
      shiftStart = 0;
      while (numToShift > 0) {
        todo = shift;
        if (todo > numToShift)
          todo = numToShift;
        dstY = shiftStart * nyPlane;
        srcY = dstY + shift * nyPlane;
        //pflush("Copying down %d\n", todo);
        if (hipMemcpy2DArrayToArray(devProj, 0, dstY, devProj, 0, srcY,
                                     sizetmp, todo * nyPlane,
                                     hipMemcpyDeviceToDevice) != hipSuccess){
          pflerr("Error copying segment of projection array down");
          numLoadedPlanes = 0;
          return 1;
        }
        numToShift -= todo;
        shiftStart += todo;
      }
    }
  }
  numLoadedPlanes = startm1;
  lsliceFirst = *lsliceStart - startm1;

  /*pflush("Initializing array num %d  first %d  loaded %d\n", numProjPlanes, 
    lsliceFirst, numLoadedPlanes); */
  // Initialize array for keeping track of copied planes, and enable copying
  for (todo = 0; todo < numProjPlanes; todo++)
    planeLoaded[todo] = todo < numLoadedPlanes ? 1 : 0;
  copyFilteredOK = 1;
  return 0;
}

// Function to load numPlanes planes of input data, starting in position
// loadStart (numbered from 1) and with starting slice number lsliceStart
int gpuloadproj(float *lines, int *numPlanes, int *lsliceStart, int *loadStart)
{
  int startm1 = *loadStart - 1;
  int todo, dstY, numCopy = 0;

  if (startm1 > 0 && checkProjLoad(numPlanes, lsliceStart, startm1)) {
    copyFilteredOK = 0;
    return 1;
  }

  // Check for valid load
  if (startm1 + *numPlanes > numProjPlanes) {
    pflush("Trying to load past end of projection array\n");
    copyFilteredOK = 0;
    numLoadedPlanes = 0;
    return 1;
  }
  
  // Find the number to copy by the last plane not already loaded
  if (copyFilteredOK) {
    for (todo = startm1; todo < startm1 + *numPlanes; todo++)
      if (!planeLoaded[todo])
        numCopy = todo + 1 - startm1;
  }
  copyFilteredOK = 0;

  // Finally do the load
  dstY = startm1 * nyPlane;
  todo = numCopy * nyPlane * nxPlane * 4;
  //if (numCopy) pflush("Loading %d planes\n", numCopy);
  if (numCopy && hipMemcpyToArray(devProj, 0, dstY, lines, todo,
                                   hipMemcpyHostToDevice) != hipSuccess) {
    pflerr("Failed to copy projection array to device");
    numLoadedPlanes = 0;
    return 1;
  }
  numLoadedPlanes = startm1 + *numPlanes;
  lsliceFirst = *lsliceStart - startm1;
  return 0;
}

// Function to do initial check on parameters in load/shift calls
static int checkProjLoad(int *numPlanes, int *lsliceStart, int startm1)
{
  if (!numLoadedPlanes) {
    pflush("Trying to load into higher planes when none are loaded\n");
    return 1;
  }
  if (lsliceFirst + numLoadedPlanes != *lsliceStart) {
    pflush("Starting slice %d does not match first slice %d + num loaded %d"
            "\n", *lsliceStart, lsliceFirst, numLoadedPlanes);
    numLoadedPlanes = 0;
    return 1;
  }
  if (startm1 > numLoadedPlanes) {
    pflush("Starting plane %d is past number loaded %d\n", startm1+1, 
           numLoadedPlanes);
    numLoadedPlanes = 0;
    return 1;
  }
  return 0;
}

/*
 * ROUTINES FOR RADIAL FILTERING OF INPUT LINES
 */

// Kernel to multiply the FFT by the filter
__global__ void filterFFT(float *FFT, float *filter, int nxprj2, int nviews, 
                          float scale)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < nviews && j < nxprj2)
    FFT[i * nxprj2 + j] *= filter[i * nxprj2 + j] * scale;
}

// Function to load the filter lines into the array and generate plans
int gpuloadfilter(float *lines)
{
  size_t sizetmp = nxPlane * numViews * numFilts * sizeof(float);
  if (hipMemcpy(radialFilt, lines, sizetmp, hipMemcpyHostToDevice) !=
      hipSuccess) {
    pflerr("Failed to copy radial filters to GPU array");
    gpudone();
    return 1;
  }
  if (hipfftPlan1d(&forwardPlan, nxPlane - 2, HIPFFT_R2C, numViews) != 
      HIPFFT_SUCCESS || hipfftPlan1d(&inversePlan, nxPlane - 2, HIPFFT_C2R, 
                                   numViews) != HIPFFT_SUCCESS) {
    pflush("Failed to generate a plan for CUFFT\n");
    gpudone();
    return 1;
  }
  return 0;
}

// Function to filter the set of input lines
int gpufilterlines(float *lines, int *lslice, int *filterSet)
{
  int ind, blockX = 16;
  size_t sizetmp = nxPlane * numViews * sizeof(float);
  float scale = 1.f / (nxPlane - 2);
  hipError_t err;
  if (hipMemcpy(devFFT, lines, sizetmp, hipMemcpyHostToDevice) !=
      hipSuccess) {
    pflerr("Failed to copy lines to GPU array for radial filtering");
    return 1;
  }
  if (hipfftExecR2C(forwardPlan, devFFT, (hipfftComplex *)devFFT) != 
      HIPFFT_SUCCESS) {
    pflush("Failure in forward FFT on GPU\n");
    return 1;
  }
  
  // Filter!!!
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((nxPlane + blockSize.x - 1) / blockSize.x, 
                (numViews + blockSize.y - 1) / blockSize.y, 1);

  filterFFT<<<gridSize, blockSize>>>
    (devFFT, radialFilt + (*filterSet - 1) * nxPlane * numViews, nxPlane, 
     numViews, scale);
  err = hipGetLastError();
  if (err != hipSuccess) {
    pflerr("Error executing threads for filtering"); 
    return 1;
  }
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after filtering");
    return 1;
  }

  if (hipfftExecC2R(inversePlan, (hipfftComplex *)devFFT, devFFT) != 
      HIPFFT_SUCCESS) {
    pflush("Failure in inverse FFT on GPU\n");
    return 1;
  }
  if (hipMemcpy(lines, devFFT, sizetmp, hipMemcpyDeviceToHost) !=
      hipSuccess) {
    pflerr("Failed to copy radial filtered lines back from GPU array");
    return 1;
  }
  
  // If copying is OK and it is a slice in needed range, copy it to proj
  if (copyFilteredOK) {
    ind = *lslice - lsliceFirst;
    if (ind >= 0 && ind < numProjPlanes) {
      //pflush("Copying %d to plane %d\n", *lslice,ind);
      if (hipMemcpyToArray(devProj, 0, ind * numViews, devFFT, sizetmp,
                            hipMemcpyDeviceToDevice) == hipSuccess)
        planeLoaded[ind] = 1;
    }
  }
  return 0;
}

/*
 * ROUTINES FOR SIMPLE BACK-PROJECTION (NO X-AXIS TILT, ETC)
 */

// Kernel for simple back-projection with testing at ends of lines
__global__ void bpNoXtTest(float *slice, int pitch, int jbase, int iwide,
                             int nxprj, int ithick, int nviews, 
                             float xcenin, float xcen, float ycen, 
                             float edgefill)
{
  float cbeta, sbeta, zpart, kproj, xp;
  float sum = 0.;
  int iv;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (j < iwide && i < ithick) {
    for (iv = 0; iv < nviews; iv++) {
      cbeta = tables[iv+COSOFS];
      sbeta = tables[iv+SINOFS];
      zpart = (i + 1 - ycen) * sbeta + xcenin;
      xp =  zpart + (j + 1 - xcen) * cbeta - 0.5f;
      //if (i == 150) printf("%d %d %d  %d  %d  %.2f  %.2f\n", iv, j, i, jlft, jrt, zpart + (1 - xcen) * cbeta - 0.5f, zpart + (nxprj - xcen) * cbeta - 0.5f);
      if (xp >= 0.5 && xp <= nxprj - 0.5) {
        kproj = iv + 0.5f;
        sum += tex2D(projtex, xp, kproj);
      } else {
        sum += edgefill;
      }
    }
    slice[i * pitch + j] = sum;
  }
}

// Kernel for simple back-projection with no testing
__global__ void bpNoXtFast(float *slice, int pitch, int jbase, int iwide,
                             int ithick, int nviews, 
                             float xcenin, float xcen, float ycen)
{
  float cbeta, sbeta, zpart, kproj, xp;
  float sum = 0.;
  int iv;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick) {
    for (iv = 0; iv < nviews; iv++) {
      cbeta = tables[iv+COSOFS];
      sbeta = tables[iv+SINOFS];
      zpart = (i + 1 - ycen) * sbeta + xcenin;
      kproj = iv + 0.5f;
      xp =  zpart + (j + 1 - xcen) * cbeta - 0.5f;
      sum += tex2D(projtex, xp, kproj);
    }
    slice[i * pitch + j] = sum;
  }
}

// Function to run simple backprojection
int gpubpnox(float *slice, float *lines, float *sbeta, float *cbeta,
             int *nxprj, float *xcenin, float *xcen, float *ycen,
             float *edgefill)
{
  size_t sizetmp = sizeof(float) * nxPlane * numViews;
  float cosinv[DELTA_OFS];
  int iv, jlft, jrt, jlftmax, jrtmin, gridLeft, gridFast, gridRight;
  float zpart, xlft, xrt, xlfttmp, xrttmp;
  int i, blockX = 16;

  if (loadBetaInvertCos(cbeta, sbeta, cosinv, numViews))
    return 1;

  // Copy projections
  if (hipMemcpyToArray(devProj, 0, 0, lines, sizetmp, hipMemcpyHostToDevice)
      != hipSuccess) {
    pflerr("Failed to copy projection array to device");
    return 1;
  }

  // Find limits of region that needs no testing
  jlftmax = 1;
  jrtmin = sliceWidth;
  for (iv = 0; iv < numViews; iv++) {
    for (i = 0; i <= sliceThick - 1; i += sliceThick - 1) {
      zpart = (i + 1 - *ycen) * sbeta[iv] + *xcenin;
      xlfttmp = (1. - zpart) * cosinv[iv] + *xcen;
      xrttmp = (*nxprj - zpart) * cosinv[iv] + *xcen;
      xlft = fmin(xlfttmp, xrttmp);
      xrt = fmax(xlfttmp, xrttmp);
      jlft = (int)ceilf(xlft);
      jrt = (int)ceilf(xrt) - 1;
      jlftmax = max(jlftmax, jlft);
      jrtmin = min(jrtmin, jrt);
      //printf("%d %f %d %.2f %d  %d  %.2f  %.2f\n", iv, cbet, i, zpart, jlft, jrt, zpart + (1 - *xcen) * cbet - 0.5f, zpart + (*nxprj - *xcen) * cbet - 0.5f);
    }
  }

  // Figure out grid sizes for left test, fast, and right test regions
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (sliceThick + blockSize.y - 1) / blockSize.y, 1);

  gridLeft = (jlftmax - 1 + blockX - 1) / blockX;
  gridFast = jrtmin / blockX - gridLeft;
  if (gridFast <= 0) {
    gridLeft = gridSize.x;
    gridRight = 0;
  } else
    gridRight = gridSize.x - (gridFast + gridLeft);

  if (gridLeft > 0) {
    gridSize.x = gridLeft;
    bpNoXtTest<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, 0, sliceWidth, *nxprj, sliceThick, 
       numViews, *xcenin, *xcen, *ycen, *edgefill);
    if (testReportErr("in left test region of backprojection"))
      return 1;
  }

  if (gridFast > 0) {
    gridSize.x = gridFast;
    bpNoXtFast<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, blockX * gridLeft, sliceWidth,
       sliceThick, numViews, *xcenin, *xcen, *ycen);
    if (testReportErr("in no-test region of backprojection"))
      return 1;
  }

  if (gridRight > 0) {
    gridSize.x = gridRight;
    bpNoXtTest<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, blockX * (gridLeft + gridFast), sliceWidth, 
       *nxprj, sliceThick, numViews, *xcenin, *xcen, *ycen, *
       edgefill);
    if (testReportErr("in right test region of backprojection"))
      return 1;
  }

  return (synchronizeCopySlice(devSlice, slicePitch, slice, sliceWidth, 
                               sliceThick));
    
}

/*
 * ROUTINES FOR BACK-PROJECTION WITH X AXIS TILT AND/OR Z FACTORS
 */

// Kernel for BP with X-axis tilt/Z-factors and testing at ends of lines
__global__ void bpXtiltTest(float *slice, int pitch, int jbase, int iwide,
                            int nxprj, int nyprj, int ithick, int nviews, 
                            float xcenin, float xcen, float ycen, float yy,
                            float slicen, int lsliceBase, float edgefill)
{
  float cbeta, sbeta, zpart, kproj, xp, zz, calpha, salpha, fj, yproj, xx;
  float sum = 0.;
  int iv, jproj;
  float ytol = 3.05f;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (j < iwide && i < ithick) {
    zz = (i + 1 - ycen);
    xx = (j + 1 - xcen);
    for (iv = 0; iv < nviews; iv++) {
      cbeta = tables[iv+COSOFS];
      sbeta = tables[iv+SINOFS];
      salpha = tables[iv+SALOFS];
      calpha = tables[iv+CALOFS];
      zpart = yy * salpha * sbeta + 
        zz * (calpha * sbeta + tables[iv+XZFOFS]) + xcenin;
      yproj = yy * calpha - zz * (salpha - tables[iv+YZFOFS]) + slicen;
      xp =  zpart + xx * cbeta - 0.5f;
      if (yproj >= 1. - ytol && yproj <= nyprj + ytol && xp >= 0.5 && 
          xp < nxprj - 0.5) {
        yproj = fmax(1.f, fmin((float)nyprj, yproj));
        jproj = min((int)yproj, nyprj - 1);
        fj = yproj - jproj;
        kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
        sum += (1.f - fj) * tex2D(projtex, xp, kproj) + 
          fj * tex2D(projtex, xp, kproj + nviews);
      } else {
        sum += edgefill;
      }
    }
    slice[i * pitch + j] = sum;
  }
}

// Kernel for BP with X-axis tilt/Z-factors and no testing 
__global__ void bpXtiltFast(float *slice, int pitch, int jbase, int iwide,
                            int ithick, int nviews, float xcenin, float xcen, 
                            float ycen, float yy, float slicen, int lsliceBase)
{
  float cbeta, sbeta, zpart, kproj, xp, zz, calpha, salpha, fj, yproj, xx;
  float sum = 0.;
  int iv, jproj;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick) {
    zz = (i + 1 - ycen);
    xx = (j + 1 - xcen);
    for (iv = 0; iv < nviews; iv++) {
      cbeta = tables[iv+COSOFS];
      sbeta = tables[iv+SINOFS];
      salpha = tables[iv+SALOFS];
      calpha = tables[iv+CALOFS];
      zpart = yy * salpha * sbeta + 
        zz * (calpha * sbeta + tables[iv+XZFOFS]) + xcenin;
      yproj = yy * calpha - zz * (salpha - tables[iv+YZFOFS]) + slicen;
      jproj = (int)yproj;
      fj = yproj - jproj;
      kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
      xp =  zpart + xx * cbeta - 0.5f;
      sum += (1.f - fj) * tex2D(projtex, xp, kproj) + 
        fj * tex2D(projtex, xp, kproj + nviews);
    }
    slice[i * pitch + j] = sum;
  }
}

// Function to run back-projection with X-axis tilt/Z-factors
int gpubpxtilt(float *slice, float *sbeta, float *cbeta, 
               float *salpha, float *calpha, float *xzfac, float *yzfac,
               int *nxprj, int *nyprj, float *xcenin, float *xcen, float *ycen,
               int *lslice, float *slicen, float *edgefill)
{
  int iv, jlft, jrt, jlftmax, jrtmin, gridLeft, gridFast, gridRight;
  float zpart, xlft, xrt, xlfttmp, xrttmp, yy, zz, yproj;
  int i, ytest, blockX = 16;
  float cosinv[DELTA_OFS];

  if (loadBetaInvertCos(cbeta, sbeta, cosinv, numViews))
    return 1;

  // Copy alphas and z factors
  iv = numViews * sizeof(float);
  if (hipMemcpyToSymbol(HIP_SYMBOL(tables), calpha, iv, CALOFS*4, hipMemcpyHostToDevice)
      || hipMemcpyToSymbol(HIP_SYMBOL(tables), salpha, iv, SALOFS*4,
                            hipMemcpyHostToDevice) ||
      hipMemcpyToSymbol(HIP_SYMBOL(tables), xzfac, iv, XZFOFS*4, hipMemcpyHostToDevice)
      || hipMemcpyToSymbol(HIP_SYMBOL(tables), yzfac, iv, YZFOFS*4,
                            hipMemcpyHostToDevice)) {
    pflerr("Failed to copy constant data to GPU");
    return 1;
  }

  // Find limits of region that needs no testing.  Test every angle top & bot
  jlftmax = 1;
  jrtmin = sliceWidth;
  yy = *lslice - *slicen;
  ytest = 0;
  for (iv = 0; iv < numViews; iv++) {
    for (i = 0; i <= sliceThick - 1; i += sliceThick - 1) {
      zz = (i + 1 - *ycen);
      zpart = yy * salpha[iv] * sbeta[iv] + zz * (calpha[iv] * sbeta[iv] +
                                                  xzfac[iv]) + *xcenin;
      yproj = yy * calpha[iv] - zz * (salpha[iv] - yzfac[iv]) + *slicen;
      if (yproj < 1 || yproj > *nyprj - 1)
        ytest = 1;
      xlfttmp = (1. - zpart) * cosinv[iv] + *xcen;
      xrttmp = (*nxprj - zpart) * cosinv[iv] + *xcen;
      xlft = fmin(xlfttmp, xrttmp);
      xrt = fmax(xlfttmp, xrttmp);
      jlft = (int)ceilf(xlft);
      jrt = (int)ceilf(xrt) - 1;
      jlftmax = max(jlftmax, jlft);
      jrtmin = min(jrtmin, jrt);
      //printf("%d %f %d %.2f %d  %d  %.2f  %.2f\n", iv, cbet, i, zpart, jlft, jrt, zpart + (1 - *xcen) * cbet - 0.5f, zpart + (*nxprj - *xcen) * cbet - 0.5f);
    }
  }

  // Figure out grid sizes for left test, fast, and right test regions
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (sliceThick + blockSize.y - 1) / blockSize.y, 1);

  gridLeft = (jlftmax - 1 + blockX - 1) / blockX;
  gridFast = jrtmin / blockX - gridLeft;
  if (gridFast <= 0 || ytest) {
    gridLeft = gridSize.x;
    gridRight = 0;
    gridFast = 0;
  } else
    gridRight = gridSize.x - (gridFast + gridLeft);

  if (gridLeft > 0) {
    gridSize.x = gridLeft;
    bpXtiltTest<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, 0, sliceWidth, *nxprj, *nyprj, sliceThick, 
       numViews, *xcenin, *xcen, *ycen, yy, *slicen, lsliceFirst, *edgefill);
    if (testReportErr("in left test region of backprojection"))
      return 1;
  }

  if (gridFast > 0) {
    gridSize.x = gridFast;
    bpXtiltFast<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, blockX * gridLeft, sliceWidth, 
       sliceThick, numViews, *xcenin, *xcen, *ycen, yy, *slicen, lsliceFirst);
    if (testReportErr("in no-test region of backprojection"))
      return 1;
  }

  if (gridRight > 0) {
    gridSize.x = gridRight;
    bpXtiltTest<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, blockX * (gridLeft + gridFast), sliceWidth, 
       *nxprj, *nyprj, sliceThick, 
       numViews, *xcenin, *xcen, *ycen, yy, *slicen, lsliceFirst, *edgefill);
    if (testReportErr("in right test region of backprojection"))
      return 1;
  }

  return (synchronizeCopySlice(devSlice, slicePitch, slice, sliceWidth,
                               sliceThick));
}

/*
 * ROUTINES FOR BACK-PROJECTION WITH LOCAL ALIGNMENTS
 */

// Kernel for back-projection using local projection factors, testing as needed
__global__ void bpLocalTest(float *slice, int slPitch, float *xprojf, 
                            float *xprojz, float *yprojf, float *yprojz, 
                            int localPitch, int iwide,
                            int nxprj, int lsliceLast, int ithick, int nviews,
                            float ycen, int lsliceBase, float edgeFill)
{
  float kproj, xp, zz, fj, yproj;
  float sum = 0.;
  float ytol = 3.05f;
  int iv, jproj, ind;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick && j < iwide) {
    zz = (i + 1 - ycen);
    for (iv = 0; iv < nviews; iv++) {
      ind = iv * localPitch + j;
      xp = xprojf[ind] + zz * xprojz[ind] - 0.5f;
      yproj = yprojf[ind] + zz * yprojz[ind];
      if (yproj >= lsliceBase - ytol && yproj <= lsliceLast + ytol && 
          xp >= 0.5f && xp < nxprj - 0.5f) {
        yproj = fmax((float)lsliceBase, fmin((float)lsliceLast, yproj));
        jproj = min((int)yproj, lsliceLast - 1);
        fj = yproj - jproj;
        kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
        sum += (1.f - fj) * tex2D(projtex, xp, kproj) + 
          fj * tex2D(projtex, xp, kproj + nviews);
      } else {
        sum += edgeFill;
      }
    }
    slice[i * slPitch + j] = sum;
  }
}

// Kernel for computing the local projection factors from warping data
__global__ void localProjFactors
(float *xprjf, float *xprjz, float *yprjf, float *yprjz, int pitch, int iv, 
 int nviews, int iwide, int minX, int lslice, int nlines, int nxwarp, 
 int nywarp, int ixswarp, int iyswarp, int idxwarp, int idywarp, float xcen,
 float xcenin, float xcenPdelxx, float slicen)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int line = blockIdx.y * blockDim.y + threadIdx.y;
  int ind1,ind2,ind3,ind4,ixc,ixt,ixpos,iyt,iypos;
  float fnd1,fnd2,fnd3,fnd4,yzf1,yzf2,yzf3,yzf4,jpos;
  float f1,f2,f3,f4,xx,yy,fx,fy;
  float calf,salf,a11,a12,a21,a22,xadd,yadd,xalladd,yalladd;
  float calf2,salf2,a112,a122,a212,a222,xadd2,yadd2;
  float calf3,salf3,a113,a123,a213,a223,xadd3,yadd3;
  float calf4,salf4,a114,a124,a214,a224,xadd4,yadd4;
  float f1x,f2x,f3x,f4x,f1xy,f2xy,f3xy,f4xy;
  float f1y,f2y,f3y,f4y,f1yy,f2yy,f3yy,f4yy;
  float xp1f,xp1z,yp1f,xp2f,xp2z,yp2f,xp3f,xp3z,yp3f,xp4f,xp4z,yp4f;
  float cbeta,sbeta,cbeta2,sbeta2,cbeta3,sbeta3,cbeta4,sbeta4;

  if (j >= iwide || line >= nlines)
    return;
  if (iv < 0)
    iv = line;
  else
    lslice += line;

  // Need to add 1 to j when it is used as a position
  jpos = j + minX + 1;
  ixc=(int)floor(jpos-xcen+xcenPdelxx+0.5f);
  ixt=min(max(ixc-ixswarp,0),(nxwarp-1)*idxwarp);
  ixpos=min(ixt/idxwarp+1,nxwarp-1);
  fx=((float)(ixt-(ixpos-1)*idxwarp))/idxwarp;
  iyt=min(max(lslice-iyswarp,0),(nywarp-1)*idywarp);
  iypos=min(iyt/idywarp+1,nywarp-1);
  fy=((float)(iyt-(iypos-1)*idywarp))/idywarp;

  ind1=(nxwarp*(iypos-1)+ixpos-1)*nviews+iv;
  ind2=ind1+nviews;
  ind3=ind1+nxwarp*nviews;
  ind4=ind3+nviews;
  f1=(1.-fy)*(1.-fx);
  f2=(1.-fy)*fx;
  f3=fy*(1.-fx);
  f4=fy*fx;
  fnd1 = ind1;
  fnd2 = ind2;
  fnd3 = ind3;
  fnd4 = ind4;
  
  cbeta=tex2D(localtex,fnd1,CBIND);
  sbeta=tex2D(localtex,fnd1,SBIND);
  calf=tex2D(localtex,fnd1,CAIND);
  salf=tex2D(localtex,fnd1,SAIND);
  a11=tex2D(localtex,fnd1,F11IND);
  a12=tex2D(localtex,fnd1,F12IND);
  a21=tex2D(localtex,fnd1,F21IND);
  a22=tex2D(localtex,fnd1,F22IND);
  xadd=tex2D(localtex,fnd1,F13IND)+xcenin-xcenin*a11-slicen*a12;
  yadd=tex2D(localtex,fnd1,F23IND)+slicen-xcenin*a21-slicen*a22;

  cbeta2=tex2D(localtex,fnd2,CBIND);
  sbeta2=tex2D(localtex,fnd2,SBIND);
  calf2=tex2D(localtex,fnd2,CAIND);
  salf2=tex2D(localtex,fnd2,SAIND);
  a112=tex2D(localtex,fnd2,F11IND);
  a122=tex2D(localtex,fnd2,F12IND);
  a212=tex2D(localtex,fnd2,F21IND);
  a222=tex2D(localtex,fnd2,F22IND);
  xadd2=tex2D(localtex,fnd2,F13IND)+xcenin-xcenin*a112-slicen*a122;
  yadd2=tex2D(localtex,fnd2,F23IND)+slicen-xcenin*a212-slicen*a222;

  cbeta3=tex2D(localtex,fnd3,CBIND);
  sbeta3=tex2D(localtex,fnd3,SBIND);
  calf3=tex2D(localtex,fnd3,CAIND);
  salf3=tex2D(localtex,fnd3,SAIND);
  a113=tex2D(localtex,fnd3,F11IND);
  a123=tex2D(localtex,fnd3,F12IND);
  a213=tex2D(localtex,fnd3,F21IND);
  a223=tex2D(localtex,fnd3,F22IND);
  xadd3=tex2D(localtex,fnd3,F13IND)+xcenin-xcenin*a113-slicen*a123;
  yadd3=tex2D(localtex,fnd3,F23IND)+slicen-xcenin*a213-slicen*a223;

  cbeta4=tex2D(localtex,fnd4,CBIND);
  sbeta4=tex2D(localtex,fnd4,SBIND);
  calf4=tex2D(localtex,fnd4,CAIND);
  salf4=tex2D(localtex,fnd4,SAIND);
  a114=tex2D(localtex,fnd4,F11IND);
  a124=tex2D(localtex,fnd4,F12IND);
  a214=tex2D(localtex,fnd4,F21IND);
  a224=tex2D(localtex,fnd4,F22IND);
  xadd4=tex2D(localtex,fnd4,F13IND)+xcenin-xcenin*a114-slicen*a124;
  yadd4=tex2D(localtex,fnd4,F23IND)+slicen-xcenin*a214-slicen*a224;
       
  f1x=f1*a11;
  f2x=f2*a112;
  f3x=f3*a113;
  f4x=f4*a114;
  f1xy=f1*a12;
  f2xy=f2*a122;
  f3xy=f3*a123;
  f4xy=f4*a124;

  f1y=f1*a21;
  f2y=f2*a212;
  f3y=f3*a213;
  f4y=f4*a214;
  f1yy=f1*a22;
  f2yy=f2*a222;
  f3yy=f3*a223;
  f4yy=f4*a224;

  xalladd=f1*xadd+f2*xadd2+f3*xadd3+f4*xadd4;
  yalladd=f1*yadd+f2*yadd2+f3*yadd3+f4*yadd4;
       
  // Each projection position is a sum of a fixed factor ("..f")
  // and a factor that multiplies z ("..z")
   
  xx=jpos-xcen;
  yy=lslice-slicen;
  xp1f=xx*cbeta + yy*salf*sbeta + xcenPdelxx;
  xp1z=calf*sbeta + tex2D(localtex,fnd1,XZFIND);
  xp2f=xx*cbeta2 + yy*salf2*sbeta2 + xcenPdelxx;
  xp2z=calf2*sbeta2 + tex2D(localtex,fnd2,XZFIND);
  xp3f=xx*cbeta3 + yy*salf3*sbeta3 + xcenPdelxx;
  xp3z=calf3*sbeta3 + tex2D(localtex,fnd3,XZFIND);
  xp4f=xx*cbeta4 + yy*salf4*sbeta4 + xcenPdelxx;
  xp4z=calf4*sbeta4 + tex2D(localtex,fnd4,XZFIND);

  yp1f=yy*calf + slicen;
  yp2f=yy*calf2 + slicen;
  yp3f=yy*calf3 + slicen;
  yp4f=yy*calf4 + slicen;

  // store the fixed and z-dependent component of the
  // projection coordinates
  yzf1 = tex2D(localtex,fnd1,YZFIND);
  yzf2 = tex2D(localtex,fnd2,YZFIND);
  yzf3 = tex2D(localtex,fnd3,YZFIND);
  yzf4 = tex2D(localtex,fnd4,YZFIND);
  ind1 = pitch * line + j;
  xprjf[ind1] =f1x*xp1f+f2x*xp2f+f3x*xp3f+f4x*xp4f+
    f1xy*yp1f+f2xy*yp2f+f3xy*yp3f+f4xy*yp4f+xalladd;
  xprjz[ind1] =f1x*xp1z+f2x*xp2z+f3x*xp3z+f4x*xp4z-
    (f1xy*(salf-yzf1)+f2xy*(salf2-yzf2)+ f3xy*(salf3-yzf3)+f4xy*(salf4-yzf4));
  yprjf[ind1] =f1y*xp1f+f2y*xp2f+f3y*xp3f+f4y*xp4f+
    f1yy*yp1f+f2yy*yp2f+f3yy*yp3f+f4yy*yp4f+yalladd;
  yprjz[ind1] =f1y*xp1z+f2y*xp2z+f3y*xp3z+f4y*xp4z-
    (f1yy*(salf-yzf1)+f2yy*(salf2-yzf2)+ f3yy*(salf3-yzf3)+f4yy*(salf4-yzf4));
}

// Function to load the local alignment data
int gpuloadlocals(float *packed, int *numWarps)
{
  size_t sizetmp = sizeof(float) * *numWarps * numViews * 12;
  if (hipMemcpyToArray(localData, 0, 0, packed, sizetmp,
                        hipMemcpyHostToDevice) != hipSuccess) {
    pflerr("Failed to copy local data to GPU array");
    gpudone();
    return 1;
  }
  return 0;
}

// Function to run back-projection with local alignments, first computing the
// the projection factors for all positions and views, then running the 
// back projection kernel
int gpubplocal(float *slice, int *lslice, int *nxwarp, int *nywarp,
               int *ixswarp, int *iyswarp, int *idxwarp, int *idywarp,
               int *nxprj, float *xcen, float *xcenin, float *delxx,
               float *ycen, float *slicen, float *edgefill)
{
  int blockX = 16;

  // Compute the local projection factors
  dim3 blockFac(blockX, 16, 1);
  dim3 gridFac((sliceWidth + blockFac.x - 1) / blockFac.x, 
                (numViews + blockFac.y - 1) / blockFac.y, 1);
  localProjFactors<<<gridFac, blockFac>>>
    (xprojf, xprojz, yprojf, yprojz, localPitch / 4, -1, numViews, sliceWidth, 
     0, *lslice, numViews, *nxwarp, *nywarp, *ixswarp, *iyswarp, *idxwarp, 
     *idywarp, *xcen, *xcenin, *xcenin+*delxx, *slicen);
  if (testReportErr("computing localProjFactors"))
      return 1;

  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after computing local factors");
    return 1;
  }

  // Do the backprojection
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (sliceThick + blockSize.y - 1) / blockSize.y, 1);

  bpLocalTest<<<gridSize, blockSize>>>
    (devSlice, slicePitch / 4, xprojf, xprojz, yprojf, yprojz, localPitch / 4,
     sliceWidth, *nxprj, lsliceFirst + numLoadedPlanes - 1, 
     sliceThick, numViews, *ycen, lsliceFirst, *edgefill);
  if (testReportErr("for local backprojection"))
      return 1;

  return (synchronizeCopySlice(devSlice, slicePitch, slice, sliceWidth, 
                               sliceThick));
}

/*
 * ROUTINES FOR REPROJECTION
 */

// Kernel to do simple reprojection (no X axis tilt or Z factors)
__global__ void reprojNox(float *lines, int pitch, int iwide, int ithick, 
                          int lsStart, int lsEnd, int lsliceBase, 
                          float xxlim, float xcenAdj, float xcenPdelxx,
                          float xprjOffset, float ycenAdj, float sbeta,
                          float cbetinv, float delz, int numz, float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, kz;
  float zz, sum, frac, zslice, xproj, xx;
  line = i + lsStart;
  sum = 0.;
  if (j >= iwide || line > lsEnd)
    return;
  for (kz = 0; kz < numz; kz++) {
    zz = 1 + kz * delz;
    frac = 1.;
    if (zz > ithick) {
      frac = 1. - (zz - (int)zz);
      zz = ithick;
    }
    zslice = zz - 0.5f;
    zz -= ycenAdj;
    xproj = j + 1 + xprjOffset;

    // the usual -0.5 is incorporated into xcenAdj
    xx = (xproj - (zz  * sbeta + xcenPdelxx)) * cbetinv + xcenAdj;
    if (xx < 0.5f || xx > xxlim) {
      sum += frac * pmean;
    } else {
      zslice += (line - lsliceBase) * ithick;
      sum += frac * tex2D(projtex, xx, zslice);
    }
  }
  lines[pitch * i + j] = sum;
}

// Kernel to do reprojection with X axis tilt and/or Z factors
__global__ void reprojXtilt(float *lines, int pitch, int iwide, int ithick, 
                            int lsStart, int lsEnd, int lsliceBase,
                            int lsliceLast, float xxlim, float xcenAdj,
                            float xcenPdelxx, float xprjOffset, float slicen,
                            float yprjOffset, float ycenAdj, float cbetinv,
                            float calfinv, float salfmyz, float salfsbet,
                            float calsbetpxz, float delz, int numz,
                            float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, kz,iys;
  float zz, sum, frac, zslice, yproj, yy, yslice, xproj, xx, fy;
  float ytol = 3.05f;
  line = i + lsStart;
  sum = 0.;
  if (j >= iwide || line > lsEnd)
    return;
  for (kz = 0; kz < numz; kz++) {
    zz = 1 + kz * delz;
    frac = 1.;
    if (zz > ithick) {
      frac = 1. - (zz - (int)zz);
      zz = ithick;
    }
    zslice = zz - 0.5f;
    zz -= ycenAdj;
    yproj = line + yprjOffset;
    yy = (yproj + zz * salfmyz - slicen) * calfinv;
    yslice = yy + slicen - yprjOffset;
    xproj = j + 1 + xprjOffset;

    // the usual -0.5 is incorporated into xcenAdj
    xx = (xproj - (yy * salfsbet + zz * calsbetpxz + xcenPdelxx)) * cbetinv +
      xcenAdj;
    if (xx < 0.5f || xx > xxlim || yslice < lsliceBase - ytol ||
        yslice > lsliceLast + ytol) {
      sum += frac * pmean;
    } else {
      iys = (int)yslice;
      if (iys < lsliceBase) {
        iys = lsliceBase;
        fy = 0.;
      } else if (iys >= lsliceLast) {
        iys = lsliceLast - 1;
        fy = 1.;
      } else {
        fy = yslice - iys;
      }
      zslice += (iys - lsliceBase) * ithick;
      sum += frac * ((1. - fy) * tex2D(projtex, xx, zslice) + 
                     fy * tex2D(projtex, xx, zslice + ithick));
    }
  }
  lines[pitch * i + j] = sum;
}

// Function to run reprojection for all cases except local alignments
int gpureproject(float *lines, float *sbeta, float *cbeta, float *salpha, 
                 float *calpha, float *xzfac, float *yzfac, float *delz,
                 int *lsStart, int *lsEnd, int *ithick,
                 float *xcen, float *xcenPdelxx, int *minXreproj, 
                 float *xprjOffset, float *ycen, int *minYreproj,
                 float *yprjOffset, float *slicen, int *ifalpha, float *pmean)
{ 
  int blockX = 16;
  int numz, numLines = *lsEnd + 1 - *lsStart;
  int lastSlice = lsliceFirst + numLoadedPlanes - 1;
  float znum, xcenAdj, salfsbet, calsbetpxz, ycenAdj, salfmyz, cbetinv,calfinv;
  znum = 1. + (*ithick - 1) / *delz;
  numz = znum;
  if (znum - numz > 0.1)
    numz++;
  xcenAdj = *xcen - (*minXreproj-1) - 0.5;
  salfsbet = *salpha * *sbeta;
  calsbetpxz = *calpha * *sbeta + *xzfac;
  ycenAdj = *ycen + 1 - *minYreproj;
  salfmyz = *salpha - *yzfac;
  cbetinv = 1. / *cbeta;
  calfinv = 1. / *calpha;

  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (numLines + blockSize.y - 1) / blockSize.y, 1);

  if (*ifalpha) {
    reprojXtilt<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, sliceWidth, *ithick, *lsStart, *lsEnd, 
       lsliceFirst, lastSlice, nxPlane - 0.5, xcenAdj, *xcenPdelxx,
       *xprjOffset, *slicen, *yprjOffset, ycenAdj, cbetinv, calfinv, salfmyz,
       salfsbet, calsbetpxz, *delz, numz, *pmean);
  } else {
    reprojNox<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, sliceWidth, *ithick, *lsStart, *lsEnd, 
       lsliceFirst, nxPlane - 0.5, xcenAdj, *xcenPdelxx, *xprjOffset,
       ycenAdj, *sbeta, cbetinv, *delz, numz, *pmean);
  }
  if (testReportErr("for reprojection"))
    return 1;
  return (synchronizeCopySlice(devSlice, slicePitch, lines, sliceWidth,
                               numLines));
}

/*
 * ROUTINES TO REPROJECT A SINGLE SLICE
 */

__global__ void reprojOneSlice(float *lines, int pitch, int iwide, int ithick, 
                               float ycen, int numproj, float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int kz;
  float zz, sum, frac, xcenAdj, xx;
  sum = 0.;
  if (j >= iwide || i >= numproj )
    return;
  for (kz = 0; kz < rpNumz[i]; kz++) {
    zz = 1 + kz * tables[COSOFS + i];
    frac = 1.;
    if (zz > ithick) {
      frac = 1. - (zz - (int)zz);
      zz = ithick;
    }
    xcenAdj = iwide / 2;

    // Invert what is multipled by sine because these sines were never inverted
    // inside tilt.f, unlike the signs for regular reproj
    // The usual 0.5 is incorporated into xcenAdj
    xx = (j + 1 - ((ycen - zz)  * tables[SINOFS+i] + xcenAdj + 0.5f)) * 
      tables[INVOFS+i] + xcenAdj;
    if (xx < 0.5f || xx > iwide - 0.5) {
      sum += frac * pmean;
    } else {
      sum += frac * tex2D(rpSlicetex, xx, zz - 0.5f);
    }
  }
  lines[pitch * i + j] = sum;
}

int gpureprojoneslice(float *slice, float *lines, float *sbeta, float *cbeta,
                      float *ycen, int *numproj, float *pmean)
{
  float znum, cosinv[DELTA_OFS];
  int numz[DELTA_OFS];
  int blockX = 16;
  int iv;

  // Get limited inverse cosines and number of points to do in Z
  loadBetaInvertCos(cbeta, sbeta, cosinv, *numproj);

  for (iv = 0; iv < *numproj; iv++) {
    znum = 1. + (sliceThick - 1) * cosinv[iv];
    numz[iv] = znum;
    if (znum - numz[iv] > 0.1)
      numz[iv]++;
  }

  // Load constant data
  iv = *numproj * sizeof(float);
  if (hipMemcpyToSymbol(HIP_SYMBOL(tables), cosinv, iv, INVOFS*4, hipMemcpyHostToDevice)
      || hipMemcpyToSymbol(HIP_SYMBOL(rpNumz), numz, iv, 0, hipMemcpyHostToDevice)) {
    pflerr("Failed to copy constant data to GPU");
    return 1;
  }

  // Copy slice
  iv = sizeof(float) * sliceWidth * sliceThick;
  if (hipMemcpyToArray(devRpSlice, 0, 0, slice, iv, hipMemcpyHostToDevice)
      != hipSuccess) {
    pflerr("Failed to copy slice array to device");
    return 1;
  }
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (*numproj + blockSize.y - 1) / blockSize.y, 1);
  reprojOneSlice<<<gridSize, blockSize>>>
    (devReproj, reprojPitch / 4, sliceWidth, sliceThick, *ycen, *numproj,
     *pmean);
  if (testReportErr("for reprojection"))
    return 1;

  return (synchronizeCopySlice(devReproj, reprojPitch, lines, nxPlane, 
                               *numproj));
}

/*
 * ROUTINES FOR REPROJECTION WITH LOCAL ALIGNMENTS
 */

/*
  Finds loaded point that projects to xproj, yproj at centered Z value
  zz, using stored values for [xy]zfac[fv].  Takes starting value in xx,yy
  and returns found value.
  Xproj, yproj are coordinates in original aligned stack.
  XX coordinate is in terms of the loaded data in X
  YY coordinate is in yterms of slices of reconstruction
*/
__device__ void loadedProjectingPoint
(float xproj, float yproj, float zz, float ofsxpz, float ofsypf, float ofsypz, 
 int nxload, int lsliceBase, int lsliceLast, float *xx, float *yy)
{
  int iter, ix, iy, ifout;
  float xp11, yp11, xp12, yp12, xp21, yp21, xerr, yerr, dypx, dxpy,dxpx;
  float dypy, den, fx, fy, findx1, findx2, findy1, findy2;

  for (iter = 0; iter < 5; iter++) {
    ix = (int)floor(*xx);
    iy = (int)floor(*yy);
    ifout = 0;
    if (ix < 1 || ix >= nxload || iy < lsliceBase || iy >= lsliceLast) {
      ifout = 1;
      ix = min(nxload - 1, max(1, ix));
      iy = min(lsliceLast - 1, max(lsliceBase, iy));
    }

    findx1 = ix - 1;
    findx2 = findx1 + 1.;
    findy1 = iy - lsliceBase;
    findy2 = findy1 + 1;
    //*yy = tex2D(pfactex, findx1, findy1 + ofsypf); return;
    xp11 = tex2D(pfactex, findx1, findy1) + 
      tex2D(pfactex, findx1, findy1 + ofsxpz) * zz;
    yp11 = tex2D(pfactex, findx1, findy1 + ofsypf) + 
      tex2D(pfactex, findx1, findy1 + ofsypz) * zz;
    xp21 = tex2D(pfactex, findx2, findy1) + 
      tex2D(pfactex, findx2, findy1 + ofsxpz) * zz;
    yp21 = tex2D(pfactex, findx2, findy1 + ofsypf) + 
      tex2D(pfactex, findx2, findy1 + ofsypz) * zz;
    xp12 = tex2D(pfactex, findx1, findy2) + 
      tex2D(pfactex, findx1, findy2 + ofsxpz) * zz;
    yp12 = tex2D(pfactex, findx1, findy2 + ofsypf) + 
      tex2D(pfactex, findx1, findy2 + ofsypz) * zz;
 
    xerr = xproj - xp11;
    yerr = yproj - yp11;
    dxpx = xp21 - xp11;
    dxpy = xp12 - xp11;
    dypx = yp21 - yp11;
    dypy = yp12 - yp11;
    den = dxpx * dypy - dxpy * dypx;
    fx = (xerr * dypy - yerr * dxpy) / den;
    fy = (dxpx * yerr - dypx * xerr) / den;
    *xx = ix + fx;
    *yy = iy + fy;
    if (fx > -0.1 & fx < 1.1 && fy > -0.1 && fy < 1.1) 
      return;
    if (ifout && (iter > 0 ||  *xx < 0. || *xx > nxload + 1 || 
                  *yy < lsliceBase - 1. || *yy > lsliceLast + 1.))
      return;
  }
}

__global__ void reprojLocal
(float *lines, int pitch, int nWarpDelz, float dxWarpDelz, int nxload,
 int iwide, int ithick, int lsStart, int lsEnd, int lsliceBase, int lsliceLast,
 float xprojMin, float xprojMax, float xcenAdj, float xcenPdelxx,
 float xprjOffset, float slicen, float yprjOffset, float ycenAdj, float cbeta,
 float sbeta, float cbetinv, float calfinv, float salfmyz, float salfsbet,
 float calsbetpxz, float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, lastZdone, iy;
  float zz, sum, frac, zslice, yproj, yy, xproj, xx, fy, zind, fline, ofsypz;
  float xxtex, ofsxpz, ofsypf;
  float ytol = 3.05f;
  float zzlim, lbaseMtol, llastPtol, dxWarpInv;
  //  int skip =390;

  line = i + lsStart;
  sum = 0.;
  if (j >= iwide || line > lsEnd)
    return;

  ofsxpz = lsliceLast + 1 - lsliceBase;
  ofsypf = ofsxpz + ofsxpz;
  ofsypz = ofsypf + ofsxpz;
  fline = i;
  yproj = line + yprjOffset;

  /* Get x projection coord, starting centered Z coordinate, and
     approximate x and y coordinates 
     X coordinate needs to be a loaded X index
     Y coordinate is in slices of reconstruction */

  // ycenAdj needs to be ycen - (minYreproj - 1)
  // xcenAdj = xcen - (minXload - 1)
  xproj = j + 1 + xprjOffset;
  zz = 1. - ycenAdj;
  yy = (yproj + zz * salfmyz - slicen) * calfinv + slicen;
  xx = (xproj - (yy*salfsbet + zz * calsbetpxz + xcenPdelxx)) * cbetinv +
    xcenAdj;
  yy -= yprjOffset;
  //lines[pitch * i + j] = yy; return;

  // Precalculate some items, doesn't help
  zzlim = ithick + 1 - ycenAdj;
  lbaseMtol = lsliceBase - ytol;
  llastPtol = lsliceLast + ytol;
  dxWarpInv = 1. / dxWarpDelz;

  // Move on ray up in Z
  lastZdone = 0;
              
  while (zz < zzlim && !lastZdone) {

    // xprojMin/Max already adjusted by 5
    if (xproj < xprojMin || xproj > xprojMax) {
      sum = sum + pmean;
      //if (zz + ycenAdj > ithick - skip) {lines[pitch * i + j] = 0; return;}
    } else {
      loadedProjectingPoint(xproj, yproj, zz, ofsxpz, ofsypf, ofsypz,
                            nxload, lsliceBase, lsliceLast, &xx, &yy);
      //if (zz + ycenAdj > ithick - skip) {lines[pitch * i + j] = yy; return;}

      // If X or Y is out of bounds, fill with mean
      if (yy < lbaseMtol || yy > llastPtol || xx < 1. || xx >= nxload) {
        sum = sum + pmean;
      } else {

        // otherwise, get x, y, z indexes, clamp y to limits, allow
        // a fractional Z pixel at top of volume
        xxtex = xx - 0.5f;
        yy = max((float)lsliceBase, min(lsliceLast - 0.01, yy));
        iy = yy;
        fy = yy - iy;
        zslice = zz + ycenAdj;
        frac = 1.;
        if (zslice > ithick) {
          frac = 1. - (zslice - (int)zslice);
          zslice = ithick - 0.5f;
          lastZdone = 1;
        } else
          zslice -= 0.5f;
                     
        // Do the interpolation
        zslice += (iy - lsliceBase) * ithick;

        sum += frac * ((1. - fy) * tex2D(projtex, xxtex, zslice) +
                       fy * tex2D(projtex, xxtex, zslice + ithick));

        // ELIMINATED JUMPING, IT TAKES 50% LONGER
      }
    }
                 
    // Adjust Z by local factor, move X approximately for next pixel
    zind = max(0., min(nWarpDelz - 1., xx * dxWarpInv));
    zz = zz + tex2D(delztex, zind, fline);
    xx = xx + sbeta;
  }
  lines[pitch * i + j] = sum;
}

int gpureprojlocal
(float *lines, float *sbeta, float *cbeta, float *salpha, float *calpha,
 float *xzfac, float *yzfac, int *nxwarp, int *nywarp, int *ixswarp, 
 int *iyswarp, int *idxwarp, int *idywarp, float *warpDelz, int *nWarpDelz, 
 float *dxWarpDelz, float *xprojMin, float *xprojMax, int *lsStart, int *lsEnd,
 int *ithick, int *iview, float *xcen, float *xcenin, float *delxx, 
 int *minXload, float *xprjOffset, float *ycenAdj, float *yprjOffset,
 float *slicen, float *pmean)
{
  int blockX = 16;
  int numLines = *lsEnd + 1 - *lsStart;
  int lastSlice = lsliceFirst + numLoadedPlanes - 1;
  int nbd, nbp;
  float xcenAdj, salfsbet, calsbetpxz, salfmyz, cbetinv,calfinv;

  xcenAdj = *xcen - (*minXload-1);
  salfsbet = *salpha * *sbeta;
  calsbetpxz = *calpha * *sbeta + *xzfac;
  salfmyz = *salpha - *yzfac;
  cbetinv = 1. / *cbeta;
  calfinv = 1. / *calpha;
  nbd = (int)floor(*yprjOffset + 0.5);

  // Compute the local projection factors
  dim3 blockFac(blockX, 16, 1);
  dim3 gridFac((nxPlane + blockFac.x - 1) / blockFac.x, 
                (numLoadedPlanes + blockFac.y - 1) / blockFac.y, 1);
  localProjFactors<<<gridFac, blockFac>>>
    (xprojf, xprojz, yprojf, yprojz, localPitch / 4, *iview - 1, numViews, 
     nxPlane, *minXload - 1, lsliceFirst + nbd, numLoadedPlanes, *nxwarp,
     *nywarp, *ixswarp,
     *iyswarp, *idxwarp, *idywarp, *xcen, *xcenin, *xcenin+*delxx, *slicen);
  if (testReportErr("computing localProjFactors"))
      return 1;
  /* return (synchronizeCopySlice(yprojf, localPitch, lines, sliceWidth,
     numLines)); */

  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after computing local factors");
    return 1;
  }

  // Load the texture arrays
  nbd = sizeof(float) * *nWarpDelz * numLines;
  nbp = sizeof(float) * nxPlane;
  if (hipMemcpyToArray(devDelz, 0, 0, warpDelz, nbd, hipMemcpyHostToDevice)
      != hipSuccess ||
      hipMemcpy2DToArray(localPfac, 0, 0, xprojf, localPitch, nbp, 
                          numLoadedPlanes, hipMemcpyDeviceToDevice) 
      != hipSuccess ||
      hipMemcpy2DToArray(localPfac, 0, numLoadedPlanes, xprojz, localPitch,
                          nbp, numLoadedPlanes, hipMemcpyDeviceToDevice) 
      != hipSuccess ||
      hipMemcpy2DToArray(localPfac, 0, 2*numLoadedPlanes, yprojf, localPitch,
                          nbp, numLoadedPlanes, hipMemcpyDeviceToDevice) 
      != hipSuccess ||
      hipMemcpy2DToArray(localPfac, 0, 3*numLoadedPlanes, yprojz, localPitch,
                          nbp, numLoadedPlanes, hipMemcpyDeviceToDevice) 
      != hipSuccess) {
    pflerr("Failed to copy local proj factors to texture array");
    return 1;
  }

  // Do the reprojection
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (numLines + blockSize.y - 1) / blockSize.y, 1);
  reprojLocal<<<gridSize, blockSize>>>
    (devSlice, slicePitch / 4, *nWarpDelz, *dxWarpDelz, nxPlane, sliceWidth,
     *ithick, *lsStart, *lsEnd, lsliceFirst, lastSlice, *xprojMin, *xprojMax,
     xcenAdj, *xcenin + *delxx, *xprjOffset, *slicen, *yprjOffset, *ycenAdj,
     *cbeta, *sbeta, cbetinv, calfinv, salfmyz, salfsbet, calsbetpxz, *pmean);
  if (testReportErr("for local reprojection"))
      return 1;
  return (synchronizeCopySlice(devSlice, slicePitch, lines, sliceWidth,
                               numLines));
}

/*
 * UTILITY ROUTINES
 */
   
// Load cosine and sine beta into constant array and compute inverse cosine
static int loadBetaInvertCos(float *cbeta, float *sbeta, float *cosinv, 
                             int num)
{
  int i, iv;
  float yy;

  // Invert cosines with limit
  for (i = 0; i < num; i++) {
    yy = cbeta[i];
    if (fabs(yy) < 0.001f)
      yy = yy >= 0 ? 0.001f : -0.001f;
    cosinv[i] = 1.f / yy;
  }

  // Copy sines/cosines
  iv = num * sizeof(float);
  if (hipMemcpyToSymbol(HIP_SYMBOL(tables), cbeta, iv, 0, hipMemcpyHostToDevice) ||
      hipMemcpyToSymbol(HIP_SYMBOL(tables), sbeta, iv, SINOFS*4,
                            hipMemcpyHostToDevice)) {
    pflerr("Failed to copy constant data to GPU");
    return 1;
  }
  return 0;
}

// Synchronize the threads and copy computed data back to caller's array
static int synchronizeCopySlice(float *devslc, int pitch, float *slice,
                                int width, int numLines)
{
  int sizetmp;
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after backprojection");
    return 1;
  }

  // Get slice back
  sizetmp = sizeof(float) * width;
  if (hipMemcpy2D(slice, sizetmp, devslc, pitch, sizetmp, numLines, 
                   hipMemcpyDeviceToHost) != hipSuccess) {
    pflerr("Error copying slice back to host");
    return 1;
  }
  return 0;
}

// Test for and report error after executing threads           
static int testReportErr(char *mess)
{
  hipError_t err;
  err = hipGetLastError();
  if (err != hipSuccess) {
    pflush("Error executing threads %s: %s\n", mess,
           hipGetErrorString(err));
    return 1;
  }
  return 0;
}

// Print a message with flushes to get it out before fortran output
static void pflush(const char *format, ...)
{
  char errorMess[512];
  va_list args;
  va_start(args, format);
  vsprintf(errorMess, format, args);
  printf("%s", errorMess);
  fflush(stdout);  
  fflush(stdout);
  va_end(args);
}

// In case of error, find the error string and print it with message
static void pflerr(const char *format, ...)
{
  hipError_t err;
  char errorMess[512];
  va_list args;
  va_start(args, format);
  vsprintf(errorMess, format, args);
  printf("%s", errorMess);
  err = hipGetLastError();
  pflush(": %s\n", hipGetErrorString(err));
  fflush(stdout);  
  fflush(stdout);
  va_end(args);
}

// Print appropriate error from allocation and free all arrays
static void allocerr(char *mess, int *nplanes, int *firstNpl,
                     int *lastNpl, int ifcuda)
{
  char *whichText[3] = {"first", "last", "only"};
  int which = 2;
  gpudone();
  if (*firstNpl != *lastNpl) {
    if (*nplanes == *firstNpl)
      which = 0;
    else if (*nplanes == *lastNpl)
      which = 1;
    else
      return;
  }
  if (ifcuda)
    pflerr("On %s try (for %d planes), %s", whichText[which], *nplanes, mess);
  else
    pflush("On %s try (for %d planes), %s", whichText[which], *nplanes, mess);
}


/*

$Log$
Revision 3.5  2010/07/26 16:31:04  mast
Changes for ncvv 3.1

Revision 3.4  2010/02/26 16:56:37  mast
Pass debug flag to gpuAvailable and return memory as a float

Revision 3.3  2010/02/22 06:04:49  mast
Added reprojection with local alignments and one-slice reprojection

Revision 3.2  2010/01/10 17:20:05  mast
Stopped selecting device more than once, setup structure to limit error
messages on repeated allocation attempts

Revision 3.1  2009/12/31 20:36:59  mast
Initial implementation


*/
