#include "hip/hip_runtime.h"
/*
 *  gpubp.cu -- Kernel and C code for CUDA-based backprojection, reprojection
 *               and Fourier filtering
 *
 *  Author: David Mastronarde   email: mast@colorado.edu
 *
 *  Copyright (C) 2009 by Boulder Laboratory for 3-Dimensional Electron
 *  Microscopy of Cells ("BL3DEMC") and the Regents of the University of 
 *  Colorado.  See dist/COPYRIGHT for full copyright notice.
 *
 *  $Id$
 *  Log at end of file
 */
#include <stdio.h>
#include <stdarg.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "imodconfig.h"

#ifdef F77FUNCAP
#define gpuavailable GPUAVAILABLE
#define gpuallocarrays GPUALLOCARRAYS
#define gpuloadproj GPULOADPROJ
#define gpushiftproj GPUSHIFTPROJ
#define gpubpxtilt GPUBPXTILT
#define gpubpnox GPUBPNOX
#define gpubplocal GPUBPLOCAL
#define gpuloadlocals GPULOADLOCALS
#define gpuloadfilter GPULOADFILTER
#define gpufilterlines GPUFILTERLINES
#define gpureproject GPUREPROJECT
#define gpudone GPUDONE
#else
#define gpuavailable gpuavailable_
#define gpuallocarrays gpuallocarrays_
#define gpuloadproj gpuloadproj_
#define gpushiftproj gpushiftproj_
#define gpubpxtilt gpubpxtilt_
#define gpubpnox gpubpnox_
#define gpubplocal gpubplocal_
#define gpuloadlocals gpuloadlocals_
#define gpuloadfilter gpuloadfilter_
#define gpufilterlines gpufilterlines_
#define gpureproject gpureproject_
#define gpudone gpudone_
#endif

#ifdef __cplusplus
extern "C" {
  int gpuavailable(int *nGPU, int *memory);
  int gpuallocarrays(int *width, int *thick, int *nxprj2, int *nviews, 
                     int *nslice, int *nxwarp);
  int gpubpnox(float *slice, float *lines, float *sbeta, float *cbeta,
               int *nxprj,
               float *xcenin, float *xcen, float *ycen, float *edgefill);
  int gpushiftproj(int *numPlanes, int *lsliceStart, int *loadStart);
  int gpuloadproj(float *lines, int *numPlanes, int *lsliceStart, 
                  int *loadStart);
  int gpubpxtilt(float *slice, float *sbeta, float *cbeta, float *salpha,
                 float *calpha, float *xzfac, float *yzfac, int *nxprj,
                 int *nyprj, float *xcenin, float *xcen, float *ycen,
                 int *lslice, float *slicen, float *edgefill);
  int gpubplocal(float *slice, int *lslice, int *nxwarp, int *nywarp,
                 int *ixswarp, int *iyswarp, int *idxwarp, int *idywarp,
                 int *nxprj, float *xcen, float *xcenin, float *delxx,
                 float *ycen, float *slicen, float *edgefill);
  int gpuloadfilter(float *lines);
  int gpuloadlocals(float *packed, int *numWarps);
  int gpufilterlines(float *lines, int *lslice);
  int gpureproject(float *lines, float *sbeta, float *cbeta, float *salpha, 
                   float *calpha, float *xzfac, float *yzfac, float *delz,
                   int *lsStart, int *lsEnd, int *ithick,
                   float *xcen, float *xcenPdelxx, int *minXreproj, 
                   float *xprjOffset, float *ycen, int *minYreproj,
                   float *yprjOffset, float *slicen, int *ifalpha, 
                   float *pmean);
  void gpudone();
}
#endif

static int checkProjLoad(int *numPlanes, int *lsliceStart, int startm1);
static int testReportErr(char *mess);
static int loadBetaInvertCos(float *cbeta, float *sbeta, float *costmp);
static int synchronizeCopySlice(float *slice, int numLines);
static void pflush(const char *format, ...);
static void pflerr(const char *format, ...);



// Offsets to positions in constant array
// For some reason 6 separate arrays did not work for xtilt case
#define DELTA_OFS  720
#define MAX_TABLE (6 * DELTA_OFS)
__constant__ float tables[MAX_TABLE];

#define COSOFS 0
#define SINOFS (1 * DELTA_OFS)
#define CALOFS (2 * DELTA_OFS)
#define SALOFS (3 * DELTA_OFS)
#define XZFOFS (4 * DELTA_OFS)
#define YZFOFS (5 * DELTA_OFS)

// Definitions for accessing the local alignments arrays with texture calls
#define F11IND 0.f
#define F21IND 1.f
#define F12IND 2.f
#define F22IND 3.f
#define F13IND 4.f
#define F23IND 5.f
#define CAIND 6.f
#define SAIND 7.f
#define CBIND 8.f
#define SBIND 9.f
#define XZFIND 10.f
#define YZFIND 11.f


// declare texture reference for 2D float textures
texture<float, 2, hipReadModeElementType> projtex;
texture<float, 2, hipReadModeElementType> localtex;

// Static variables for device arrays
static float* devSlice;
static hipArray* devProj;
static float *xprojf = NULL;
static float *xprojz = NULL;
static float *yprojf = NULL;
static float *yprojz = NULL;
static hipArray *localData = NULL;
static float *radialFilt = NULL;
static float *devFFT = NULL;

// Other static variables
static hipfftHandle forwardPlan = 0, inversePlan = 0;
static int max_gflops_device = -1;
static size_t slicePitch;
static int sliceThick, sliceWidth, numViews, numProjPlanes;
static int lsliceFirst, numLoadedPlanes, nxPlane;
static int copyFilteredOK = 0;
static int *planeLoaded;

/*
 *  SETUP/SHUTDOWN ROUTINES
 */

/*
 * Test whether a GPU is available, either a GPU of the given number if nGPU is
 * > 0, or the one with the best processing rate if nGPU is 0, and return the
 * memory in bytes.  Return value is 1 for success, 0 for failure.
 */
int gpuavailable(int *nGPU, int *memory)
{
  int current_device = 0;
  int device_count = 0;
  float gflops;
  struct hipDeviceProp_t device_properties, best_properties;
  float max_gflops = 0;
  *memory = 0;
  hipGetDeviceCount( &device_count );
  if (*nGPU != 0) {
    if (*nGPU < 0 || *nGPU > device_count)
      return 0;
    current_device = *nGPU - 1;
    device_count = *nGPU;
  }
  for (; current_device < device_count; current_device++) {
    if (hipGetDeviceProperties( &device_properties, current_device)
        != hipSuccess) {
      pflerr("Error returned from trying to get properties of GPU device %d",
               current_device);
      return 0;
    }
    gflops = device_properties.multiProcessorCount * 
      device_properties.clockRate;
    if( gflops > max_gflops ) {
      max_gflops = gflops;
      max_gflops_device = current_device;
      best_properties = device_properties;
    }
  }
    
  if (max_gflops_device >= 0) {
    *memory = best_properties.totalGlobalMem;
    return 1;
  }
  return 0;
}

/*
 * Allocate all needed arrays on the GPU.  Allocate a reconstructed slice or
 * reprojected line array of width x thick, an array for nplanes of input data 
 * each with nviews lines of length nxprj2, and local alignment arrays if
 * numWarps > 0.  If numWarp >= 0, also allocate arrays for line filtering.
 */
int gpuallocarrays(int *width, int *thick, int *nxprj2, int *nviews, 
                   int *nplanes, int *numWarps)
{
  size_t pitch1, pitch2, pitch3, pitch4, memTot;

  if (max_gflops_device < 0)
    return 1;
  if (hipSetDevice(max_gflops_device) != hipSuccess) {
    pflerr("Error selecting GPU device");
    return 1;
  }

  // Allocate memory for slice on device
  size_t sizetmp = *width * sizeof(float);
  if (hipMallocPitch((void **)&devSlice, &slicePitch, sizetmp, *thick) != 
      hipSuccess) {
    pflerr("Failed to allocate slice array on GPU device");
    return 1;
  }

  // Allocate memory for projection lines
  hipChannelFormatDesc projDesc = hipCreateChannelDesc
    (32, 0, 0, 0, hipChannelFormatKindFloat);
  if (hipMallocArray(&devProj, &projDesc, *nxprj2, *nviews * *nplanes)
      != hipSuccess) {
    pflerr("Failed to allocate projection array on GPU device");
    hipFree(devSlice);
    return 1;
  }
  memTot = sizetmp * *thick + 4 * *nxprj2 * *nviews * *nplanes;

  // set texture parameters
  projtex.addressMode[0] = hipAddressModeClamp;
  projtex.addressMode[1] = hipAddressModeClamp;
  projtex.filterMode = hipFilterModeLinear;
  projtex.normalized = false;
  
  // Bind the array to the texture
  if (hipBindTextureToArray(projtex, devProj, projDesc) != hipSuccess) {
    pflerr("Failed to bind projection array to texture");
    hipFree(devSlice);
    hipFreeArray(devProj);
    return 1;
  }

  if (*nplanes > 1) {
    planeLoaded = (int *)malloc(*nplanes * sizeof(int));
    if (!planeLoaded) {
      pflush("Failed to malloc little array planeLoaded\n");
      gpudone();
      return 1;
    }
  }

  // Get arrays for local proj factors
  if (*numWarps > 0) {
    if (hipMallocPitch((void **)&xprojf, &pitch1, sizetmp, *nviews) != 
        hipSuccess ||
        hipMallocPitch((void **)&xprojz, &pitch2, sizetmp, *nviews) != 
        hipSuccess ||
        hipMallocPitch((void **)&yprojf, &pitch3, sizetmp, *nviews) != 
        hipSuccess ||
        hipMallocPitch((void **)&yprojz, &pitch4, sizetmp, *nviews) != 
        hipSuccess  ||
        hipMallocArray(&localData, &projDesc, *numWarps * *nviews, 12) 
        != hipSuccess) {
      pflerr("Failed to allocate local factor arrays on GPU device");
      gpudone();
      return 1;
    }
    if (pitch2 != pitch1 || pitch3 != pitch1 || pitch4 != pitch1 || 
        pitch1 != slicePitch) {
      pflush("Array pitches for GPU arrays do NOT match\n");
      gpudone();
      return 1;
    }

    localtex.filterMode = hipFilterModePoint;
    localtex.normalized = false;
    if (hipBindTextureToArray(localtex, localData, projDesc) != hipSuccess) {
      pflerr("Failed to bind local factor arrays to texture");
      gpudone();
      return 1;
    }
    memTot += 4 * sizetmp * *nviews + 48 * *numWarps * *nviews;
  }

  // Get arrays for radial filtering
  if (*numWarps >= 0) {
    sizetmp = *nxprj2 * *nviews * sizeof(float);
    if (hipMalloc((void **)&devFFT, sizetmp)  != hipSuccess ||
        hipMalloc((void **)&radialFilt, sizetmp)  != hipSuccess) {
      pflerr("Failed to  allocate GPU arrays for radial filtering");
      gpudone();
      return 1;
    }
    memTot += 2 * sizetmp;
  }

  pflush("Allocated %4d MB for arrays on the GPU\n", 
         (memTot + 512*1024)/(1024*1024));
  sliceWidth = *width;
  sliceThick = *thick;
  numViews = *nviews;
  numProjPlanes = *nplanes;
  nxPlane = *nxprj2;
  return 0;
}

// Routine to free all allocated resources
void gpudone()
{
  hipFree(devSlice);
  hipFreeArray(devProj);
  hipFree(xprojf);
  hipFree(xprojz);
  hipFree(yprojf);
  hipFree(yprojz);
  hipFreeArray(localData);
  hipFree(devFFT);
  hipFree(radialFilt);
  if (forwardPlan)
    hipfftDestroy(forwardPlan);
  if (inversePlan)
    hipfftDestroy(inversePlan);
}

/*
 * ROUTINES FOR LOADING/MAINTAINING STACK OF PLANES ON GPU
 */ 

// Function to shift existing data in preparation for loading new data starting
// in position loadStart (numbered from 1) and with starting slice number
// lsliceStart
int gpushiftproj(int *numPlanes, int *lsliceStart, int *loadStart)
{
  int startm1 = *loadStart - 1;
  int shift, shiftStart, numToShift, todo, dstY, srcY;
  size_t sizetmp = nxPlane * sizeof(float);
  if (startm1 > 0) {
    if (checkProjLoad(numPlanes, lsliceStart, startm1))
      return 1;

    // Copy data down without overlap if it goes into occupied planes
    if (startm1 < numLoadedPlanes) {
      shift = numLoadedPlanes - startm1;
      numToShift = startm1;
      shiftStart = 0;
      while (numToShift > 0) {
        todo = shift;
        if (todo > numToShift)
          todo = numToShift;
        dstY = shiftStart * numViews;
        srcY = dstY + shift * numViews;
        //pflush("Copying down %d\n", todo);
        if (hipMemcpy2DArrayToArray(devProj, 0, dstY, devProj, 0, srcY,
                                     sizetmp, todo * numViews,
                                     hipMemcpyDeviceToDevice) != hipSuccess){
          pflerr("Error copying segment of projection array down");
          numLoadedPlanes = 0;
          return 1;
        }
        numToShift -= todo;
        shiftStart += todo;
      }
    }
  }
  numLoadedPlanes = startm1;
  lsliceFirst = *lsliceStart - startm1;

  /*pflush("Initializing array num %d  first %d  loaded %d\n", numProjPlanes, 
    lsliceFirst, numLoadedPlanes); */
  // Initialize array for keeping track of copied planes, and enable copying
  for (todo = 0; todo < numProjPlanes; todo++)
    planeLoaded[todo] = todo < numLoadedPlanes ? 1 : 0;
  copyFilteredOK = 1;
  return 0;
}

// Function to load numPlanes planes of input data, starting in position
// loadStart (numbered from 1) and with starting slice number lsliceStart
int gpuloadproj(float *lines, int *numPlanes, int *lsliceStart, int *loadStart)
{
  int startm1 = *loadStart - 1;
  int todo, dstY, numCopy = 0;

  if (startm1 > 0 && checkProjLoad(numPlanes, lsliceStart, startm1)) {
    copyFilteredOK = 0;
    return 1;
  }

  // Check for valid load
  if (startm1 + *numPlanes > numProjPlanes) {
    pflush("Trying to load past end of projection array\n");
    copyFilteredOK = 0;
    numLoadedPlanes = 0;
    return 1;
  }
  
  // Find the number to copy by the last plane not already loaded
  if (copyFilteredOK) {
    for (todo = startm1; todo < startm1 + *numPlanes; todo++)
      if (!planeLoaded[todo])
        numCopy = todo + 1 - startm1;
  }
  copyFilteredOK = 0;

  // Finally do the load
  dstY = startm1 * numViews;
  todo = numCopy * numViews * nxPlane * 4;
  //if (numCopy) pflush("Loading %d planes\n", numCopy);
  if (numCopy && hipMemcpyToArray(devProj, 0, dstY, lines, todo,
                                   hipMemcpyHostToDevice) != hipSuccess) {
    pflerr("Failed to copy projection array to device");
    numLoadedPlanes = 0;
    return 1;
  }
  numLoadedPlanes = startm1 + *numPlanes;
  lsliceFirst = *lsliceStart - startm1;
  return 0;
}

// Function to do initial check on parameters in load/shift calls
static int checkProjLoad(int *numPlanes, int *lsliceStart, int startm1)
{
  if (!numLoadedPlanes) {
    pflush("Trying to load into higher planes when none are loaded\n");
    return 1;
  }
  if (lsliceFirst + numLoadedPlanes != *lsliceStart) {
    pflush("Starting slice %d does not match first slice %d + num loaded %d"
            "\n", *lsliceStart, lsliceFirst, numLoadedPlanes);
    numLoadedPlanes = 0;
    return 1;
  }
  if (startm1 > numLoadedPlanes) {
    pflush("Starting plane %d is past number loaded %d\n", startm1+1, 
           numLoadedPlanes);
    numLoadedPlanes = 0;
    return 1;
  }
  return 0;
}

/*
 * ROUTINES FOR RADIAL FILTERING OF INPUT LINES
 */

// Kernel to multiply the FFT by the filter
__global__ void filterFFT(float *FFT, float *filter, int nxprj2, int nviews, 
                          float scale)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < nviews && j < nxprj2)
    FFT[i * nxprj2 + j] *= filter[i * nxprj2 + j] * scale;
}

// Function to load the filter lines into the array and generate plans
int gpuloadfilter(float *lines)
{
  size_t sizetmp = nxPlane * numViews * sizeof(float);
  if (hipMemcpy(radialFilt, lines, sizetmp, hipMemcpyHostToDevice) !=
      hipSuccess) {
    pflerr("Failed to copy radial filters to GPU array");
    gpudone();
    return 1;
  }
  if (hipfftPlan1d(&forwardPlan, nxPlane - 2, HIPFFT_R2C, numViews) != 
      HIPFFT_SUCCESS || hipfftPlan1d(&inversePlan, nxPlane - 2, HIPFFT_C2R, 
                                   numViews) != HIPFFT_SUCCESS) {
    pflush("Failed to generate a plan for CUFFT\n");
    gpudone();
    return 1;
  }
  return 0;
}

// Function to filter the set of input lines
int gpufilterlines(float *lines, int *lslice)
{
  int ind, blockX = 16;
  size_t sizetmp = nxPlane * numViews * sizeof(float);
  float scale = 1.f / (nxPlane - 2);
  hipError_t err;
  if (hipMemcpy(devFFT, lines, sizetmp, hipMemcpyHostToDevice) !=
      hipSuccess) {
    pflerr("Failed to copy lines to GPU array for radial filtering");
    return 1;
  }
  if (hipfftExecR2C(forwardPlan, devFFT, (hipfftComplex *)devFFT) != 
      HIPFFT_SUCCESS) {
    pflush("Failure in forward FFT on GPU\n");
    return 1;
  }
  
  // Filter!!!
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((nxPlane + blockSize.x - 1) / blockSize.x, 
                (numViews + blockSize.y - 1) / blockSize.y, 1);

  filterFFT<<<gridSize, blockSize>>>
    (devFFT, radialFilt, nxPlane, numViews, scale);
  err = hipGetLastError();
  if (err != hipSuccess) {
    pflerr("Error executing threads for filtering"); 
    return 1;
  }
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after filtering");
    return 1;
  }

  if (hipfftExecC2R(inversePlan, (hipfftComplex *)devFFT, devFFT) != 
      HIPFFT_SUCCESS) {
    pflush("Failure in inverse FFT on GPU\n");
    return 1;
  }
  if (hipMemcpy(lines, devFFT, sizetmp, hipMemcpyDeviceToHost) !=
      hipSuccess) {
    pflerr("Failed to copy radial filtered lines back from GPU array");
    return 1;
  }
  
  // If copying is OK and it is a slice in needed range, copy it to proj
  if (copyFilteredOK) {
    ind = *lslice - lsliceFirst;
    if (ind >= 0 && ind < numProjPlanes) {
      //pflush("Copying %d to plane %d\n", *lslice,ind);
      if (hipMemcpyToArray(devProj, 0, ind * numViews, devFFT, sizetmp,
                            hipMemcpyDeviceToDevice) == hipSuccess)
        planeLoaded[ind] = 1;
    }
  }
  return 0;
}

/*
 * ROUTINES FOR SIMPLE BACK-PROJECTION (NO X-AXIS TILT, ETC)
 */

// Kernel for simple back-projection with testing at ends of lines
__global__ void bpNoXtTest(float *slice, int pitch, int jbase, int iwide,
                             int nxprj, int ithick, int nviews, 
                             float xcenin, float xcen, float ycen, 
                             float edgefill)
{
  float cbeta, sbeta, zpart, kproj, xp;
  float sum = 0.;
  int iv;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (j < iwide && i < ithick) {
    for (iv = 0; iv < nviews; iv++) {
      cbeta = tables[iv+COSOFS];
      sbeta = tables[iv+SINOFS];
      zpart = (i + 1 - ycen) * sbeta + xcenin;
      xp =  zpart + (j + 1 - xcen) * cbeta - 0.5f;
      //if (i == 150) printf("%d %d %d  %d  %d  %.2f  %.2f\n", iv, j, i, jlft, jrt, zpart + (1 - xcen) * cbeta - 0.5f, zpart + (nxprj - xcen) * cbeta - 0.5f);
      if (xp >= 0.5 && xp <= nxprj - 0.5) {
        kproj = iv + 0.5f;
        sum += tex2D(projtex, xp, kproj);
      } else {
        sum += edgefill;
      }
    }
    slice[i * pitch + j] = sum;
  }
}

// Kernel for simple back-projection with no testing
__global__ void bpNoXtFast(float *slice, int pitch, int jbase, int iwide,
                             int ithick, int nviews, 
                             float xcenin, float xcen, float ycen)
{
  float cbeta, sbeta, zpart, kproj, xp;
  float sum = 0.;
  int iv;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick) {
    for (iv = 0; iv < nviews; iv++) {
      cbeta = tables[iv+COSOFS];
      sbeta = tables[iv+SINOFS];
      zpart = (i + 1 - ycen) * sbeta + xcenin;
      kproj = iv + 0.5f;
      xp =  zpart + (j + 1 - xcen) * cbeta - 0.5f;
      sum += tex2D(projtex, xp, kproj);
    }
    slice[i * pitch + j] = sum;
  }
}

// Function to run simple backprojection
int gpubpnox(float *slice, float *lines, float *sbeta, float *cbeta,
             int *nxprj, float *xcenin, float *xcen, float *ycen,
             float *edgefill)
{
  size_t sizetmp = sizeof(float) * nxPlane * numViews;
  float cosinv[DELTA_OFS];
  int iv, jlft, jrt, jlftmax, jrtmin, gridLeft, gridFast, gridRight;
  float zpart, xlft, xrt, xlfttmp, xrttmp;
  int i, blockX = 16;

  if (loadBetaInvertCos(cbeta, sbeta, cosinv))
    return 1;

  // Copy projections
  if (hipMemcpyToArray(devProj, 0, 0, lines, sizetmp, hipMemcpyHostToDevice)
      != hipSuccess) {
    pflerr("Failed to copy projection array to device");
    return 1;
  }

  // Find limits of region that needs no testing
  jlftmax = 1;
  jrtmin = sliceWidth;
  for (iv = 0; iv < numViews; iv++) {
    for (i = 0; i <= sliceThick - 1; i += sliceThick - 1) {
      zpart = (i + 1 - *ycen) * sbeta[iv] + *xcenin;
      xlfttmp = (1. - zpart) * cosinv[iv] + *xcen;
      xrttmp = (*nxprj - zpart) * cosinv[iv] + *xcen;
      xlft = fmin(xlfttmp, xrttmp);
      xrt = fmax(xlfttmp, xrttmp);
      jlft = (int)ceilf(xlft);
      jrt = (int)ceilf(xrt) - 1;
      jlftmax = max(jlftmax, jlft);
      jrtmin = min(jrtmin, jrt);
      //printf("%d %f %d %.2f %d  %d  %.2f  %.2f\n", iv, cbet, i, zpart, jlft, jrt, zpart + (1 - *xcen) * cbet - 0.5f, zpart + (*nxprj - *xcen) * cbet - 0.5f);
    }
  }

  // Figure out grid sizes for left test, fast, and right test regions
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (sliceThick + blockSize.y - 1) / blockSize.y, 1);

  gridLeft = (jlftmax - 1 + blockX - 1) / blockX;
  gridFast = jrtmin / blockX - gridLeft;
  if (gridFast <= 0) {
    gridLeft = gridSize.x;
    gridRight = 0;
  } else
    gridRight = gridSize.x - (gridFast + gridLeft);

  if (gridLeft > 0) {
    gridSize.x = gridLeft;
    bpNoXtTest<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, 0, sliceWidth, *nxprj, sliceThick, 
       numViews, *xcenin, *xcen, *ycen, *edgefill);
    if (testReportErr("in left test region of backprojection"))
      return 1;
  }

  if (gridFast > 0) {
    gridSize.x = gridFast;
    bpNoXtFast<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, blockX * gridLeft, sliceWidth,
       sliceThick, numViews, *xcenin, *xcen, *ycen);
    if (testReportErr("in no-test region of backprojection"))
      return 1;
  }

  if (gridRight > 0) {
    gridSize.x = gridRight;
    bpNoXtTest<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, blockX * (gridLeft + gridFast), sliceWidth, 
       *nxprj, sliceThick, numViews, *xcenin, *xcen, *ycen, *
       edgefill);
    if (testReportErr("in right test region of backprojection"))
      return 1;
  }

  return (synchronizeCopySlice(slice, sliceThick));
    
}

/*
 * ROUTINES FOR BACK-PROJECTION WITH X AXIS TILT AND/OR Z FACTORS
 */

// Kernel for BP with X-axis tilt/Z-factors and testing at ends of lines
__global__ void bpXtiltTest(float *slice, int pitch, int jbase, int iwide,
                            int nxprj, int nyprj, int ithick, int nviews, 
                            float xcenin, float xcen, float ycen, float yy,
                            float slicen, int lsliceBase, float edgefill)
{
  float cbeta, sbeta, zpart, kproj, xp, zz, calpha, salpha, fj, yproj, xx;
  float sum = 0.;
  int iv, jproj;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (j < iwide && i < ithick) {
    zz = (i + 1 - ycen);
    xx = (j + 1 - xcen);
    for (iv = 0; iv < nviews; iv++) {
      cbeta = tables[iv+COSOFS];
      sbeta = tables[iv+SINOFS];
      salpha = tables[iv+SALOFS];
      calpha = tables[iv+CALOFS];
      zpart = yy * salpha * sbeta + 
        zz * (calpha * sbeta + tables[iv+XZFOFS]) + xcenin;
      yproj = yy * calpha - zz * (salpha - tables[iv+YZFOFS]) + slicen;
      xp =  zpart + xx * cbeta - 0.5f;
      if (yproj >= 1. && yproj <= nyprj && xp >= 0.5 && xp < nxprj - 0.5) {
        jproj = min((int)yproj, nyprj - 1);
        fj = yproj - jproj;
        kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
        sum += (1.f - fj) * tex2D(projtex, xp, kproj) + 
          fj * tex2D(projtex, xp, kproj + nviews);
      } else {
        sum += edgefill;
      }
    }
    slice[i * pitch + j] = sum;
  }
}

// Kernel for BP with X-axis tilt/Z-factors and no testing 
__global__ void bpXtiltFast(float *slice, int pitch, int jbase, int iwide,
                            int ithick, int nviews, float xcenin, float xcen, 
                            float ycen, float yy, float slicen, int lsliceBase)
{
  float cbeta, sbeta, zpart, kproj, xp, zz, calpha, salpha, fj, yproj, xx;
  float sum = 0.;
  int iv, jproj;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick) {
    zz = (i + 1 - ycen);
    xx = (j + 1 - xcen);
    for (iv = 0; iv < nviews; iv++) {
      cbeta = tables[iv+COSOFS];
      sbeta = tables[iv+SINOFS];
      salpha = tables[iv+SALOFS];
      calpha = tables[iv+CALOFS];
      zpart = yy * salpha * sbeta + 
        zz * (calpha * sbeta + tables[iv+XZFOFS]) + xcenin;
      yproj = yy * calpha - zz * (salpha - tables[iv+YZFOFS]) + slicen;
      jproj = (int)yproj;
      fj = yproj - jproj;
      kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
      xp =  zpart + xx * cbeta - 0.5f;
      sum += (1.f - fj) * tex2D(projtex, xp, kproj) + 
        fj * tex2D(projtex, xp, kproj + nviews);
    }
    slice[i * pitch + j] = sum;
  }
}

// Function to run back-projection with X-axis tilt/Z-factors
int gpubpxtilt(float *slice, float *sbeta, float *cbeta, 
               float *salpha, float *calpha, float *xzfac, float *yzfac,
               int *nxprj, int *nyprj, float *xcenin, float *xcen, float *ycen,
               int *lslice, float *slicen, float *edgefill)
{
  int iv, jlft, jrt, jlftmax, jrtmin, gridLeft, gridFast, gridRight;
  float zpart, xlft, xrt, xlfttmp, xrttmp, yy, zz, yproj;
  int i, ytest, blockX = 16;
  float cosinv[DELTA_OFS];

  if (loadBetaInvertCos(cbeta, sbeta, cosinv))
    return 1;

  // Copy alphas and z factors
  iv = numViews * sizeof(float);
  if (hipMemcpyToSymbol(HIP_SYMBOL(tables), calpha, iv, CALOFS*4, hipMemcpyHostToDevice)
      || hipMemcpyToSymbol(HIP_SYMBOL(tables), salpha, iv, SALOFS*4,
                            hipMemcpyHostToDevice) ||
      hipMemcpyToSymbol(HIP_SYMBOL(tables), xzfac, iv, XZFOFS*4, hipMemcpyHostToDevice)
      || hipMemcpyToSymbol(HIP_SYMBOL(tables), yzfac, iv, YZFOFS*4,
                            hipMemcpyHostToDevice)) {
    pflerr("Failed to copy constant data to GPU");
    return 1;
  }

  // Find limits of region that needs no testing.  Test every angle top & bot
  jlftmax = 1;
  jrtmin = sliceWidth;
  yy = *lslice - *slicen;
  ytest = 0;
  for (iv = 0; iv < numViews; iv++) {
    for (i = 0; i <= sliceThick - 1; i += sliceThick - 1) {
      zz = (i + 1 - *ycen);
      zpart = yy * salpha[iv] * sbeta[iv] + zz * (calpha[iv] * sbeta[iv] +
                                                  xzfac[iv]) + *xcenin;
      yproj = yy * calpha[iv] - zz * (salpha[iv] - yzfac[iv]) + *slicen;
      if (yproj < 1 || yproj > *nyprj - 1)
        ytest = 1;
      xlfttmp = (1. - zpart) * cosinv[iv] + *xcen;
      xrttmp = (*nxprj - zpart) * cosinv[iv] + *xcen;
      xlft = fmin(xlfttmp, xrttmp);
      xrt = fmax(xlfttmp, xrttmp);
      jlft = (int)ceilf(xlft);
      jrt = (int)ceilf(xrt) - 1;
      jlftmax = max(jlftmax, jlft);
      jrtmin = min(jrtmin, jrt);
      //printf("%d %f %d %.2f %d  %d  %.2f  %.2f\n", iv, cbet, i, zpart, jlft, jrt, zpart + (1 - *xcen) * cbet - 0.5f, zpart + (*nxprj - *xcen) * cbet - 0.5f);
    }
  }

  // Figure out grid sizes for left test, fast, and right test regions
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (sliceThick + blockSize.y - 1) / blockSize.y, 1);

  gridLeft = (jlftmax - 1 + blockX - 1) / blockX;
  gridFast = jrtmin / blockX - gridLeft;
  if (gridFast <= 0 || ytest) {
    gridLeft = gridSize.x;
    gridRight = 0;
    gridFast = 0;
  } else
    gridRight = gridSize.x - (gridFast + gridLeft);

  if (gridLeft > 0) {
    gridSize.x = gridLeft;
    bpXtiltTest<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, 0, sliceWidth, *nxprj, *nyprj, sliceThick, 
       numViews, *xcenin, *xcen, *ycen, yy, *slicen, lsliceFirst, *edgefill);
    if (testReportErr("in left test region of backprojection"))
      return 1;
  }

  if (gridFast > 0) {
    gridSize.x = gridFast;
    bpXtiltFast<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, blockX * gridLeft, sliceWidth, 
       sliceThick, numViews, *xcenin, *xcen, *ycen, yy, *slicen, lsliceFirst);
    if (testReportErr("in no-test region of backprojection"))
      return 1;
  }

  if (gridRight > 0) {
    gridSize.x = gridRight;
    bpXtiltTest<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, blockX * (gridLeft + gridFast), sliceWidth, 
       *nxprj, *nyprj, sliceThick, 
       numViews, *xcenin, *xcen, *ycen, yy, *slicen, lsliceFirst, *edgefill);
    if (testReportErr("in right test region of backprojection"))
      return 1;
  }

  return (synchronizeCopySlice(slice, sliceThick));
}

/*
 * ROUTINES FOR BACK-PROJECTION WITH LOCAL ALIGNMENTS
 */

// Kernel for back-projection using local projection factors, testing as needed
__global__ void bpLocalTest(float *slice, float *xprojf, float *xprojz, 
                            float *yprojf, float *yprojz, int pitch, int iwide,
                            int nxprj, int lsliceLast, int ithick, int nviews,
                            float ycen, int lsliceBase, float edgeFill)
{
  float kproj, xp, zz, fj, yproj;
  float sum = 0.;
  int iv, jproj, ind;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick && j < iwide) {
    zz = (i + 1 - ycen);
    for (iv = 0; iv < nviews; iv++) {
      ind = iv * pitch + j;
      xp = xprojf[ind] + zz * xprojz[ind] - 0.5f;
      yproj = yprojf[ind] + zz * yprojz[ind];
      if (yproj >= lsliceBase && yproj <= lsliceLast && xp >= 0.5 && 
          xp < nxprj - 0.5) {
        jproj = min((int)yproj, lsliceLast - 1);
        fj = yproj - jproj;
        kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
        sum += (1.f - fj) * tex2D(projtex, xp, kproj) + 
          fj * tex2D(projtex, xp, kproj + nviews);
      } else {
        sum += edgeFill;
      }
    }
    slice[i * pitch + j] = sum;
  }
}

// Kernel for computing the local projection factors from warping data
__global__ void localProjFactors
(float *xprjf, float *xprjz, float *yprjf, float *yprjz, int pitch, int nviews,
 int iwide, int lslice, int nxwarp, int nywarp, int ixswarp, int iyswarp,
 int idxwarp, int idywarp, float xcen, float xcenin, float xcenPdelxx,
 float slicen)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int iv = blockIdx.y * blockDim.y + threadIdx.y;
  int ind1,ind2,ind3,ind4,ixc,ixt,ixpos,iyt,iypos;
  float fnd1,fnd2,fnd3,fnd4,yzf1,yzf2,yzf3,yzf4;
  float f1,f2,f3,f4,xx,yy,fx,fy;
  float calf,salf,a11,a12,a21,a22,xadd,yadd,xalladd,yalladd;
  float calf2,salf2,a112,a122,a212,a222,xadd2,yadd2;
  float calf3,salf3,a113,a123,a213,a223,xadd3,yadd3;
  float calf4,salf4,a114,a124,a214,a224,xadd4,yadd4;
  float f1x,f2x,f3x,f4x,f1xy,f2xy,f3xy,f4xy;
  float f1y,f2y,f3y,f4y,f1yy,f2yy,f3yy,f4yy;
  float xp1f,xp1z,yp1f,xp2f,xp2z,yp2f,xp3f,xp3z,yp3f,xp4f,xp4z,yp4f;
  float cbeta,sbeta,cbeta2,sbeta2,cbeta3,sbeta3,cbeta4,sbeta4;

  if (j >= iwide || iv >= nviews)
    return;

  // Need to add 1 to j when it is used as a position
  ixc=(int)floor(j-xcen+xcenPdelxx+1.5f);
  ixt=min(max(ixc-ixswarp,0),(nxwarp-1)*idxwarp);
  ixpos=min(ixt/idxwarp+1,nxwarp-1);
  fx=((float)(ixt-(ixpos-1)*idxwarp))/idxwarp;
  iyt=min(max(lslice-iyswarp,0),(nywarp-1)*idywarp);
  iypos=min(iyt/idywarp+1,nywarp-1);
  fy=((float)(iyt-(iypos-1)*idywarp))/idywarp;

  ind1=(nxwarp*(iypos-1)+ixpos-1)*nviews+iv;
  ind2=ind1+nviews;
  ind3=ind1+nxwarp*nviews;
  ind4=ind3+nviews;
  f1=(1.-fy)*(1.-fx);
  f2=(1.-fy)*fx;
  f3=fy*(1.-fx);
  f4=fy*fx;
  fnd1 = ind1;
  fnd2 = ind2;
  fnd3 = ind3;
  fnd4 = ind4;
  
  cbeta=tex2D(localtex,fnd1,CBIND);
  sbeta=tex2D(localtex,fnd1,SBIND);
  calf=tex2D(localtex,fnd1,CAIND);
  salf=tex2D(localtex,fnd1,SAIND);
  a11=tex2D(localtex,fnd1,F11IND);
  a12=tex2D(localtex,fnd1,F12IND);
  a21=tex2D(localtex,fnd1,F21IND);
  a22=tex2D(localtex,fnd1,F22IND);
  xadd=tex2D(localtex,fnd1,F13IND)+xcenin-xcenin*a11-slicen*a12;
  yadd=tex2D(localtex,fnd1,F23IND)+slicen-xcenin*a21-slicen*a22;

  cbeta2=tex2D(localtex,fnd2,CBIND);
  sbeta2=tex2D(localtex,fnd2,SBIND);
  calf2=tex2D(localtex,fnd2,CAIND);
  salf2=tex2D(localtex,fnd2,SAIND);
  a112=tex2D(localtex,fnd2,F11IND);
  a122=tex2D(localtex,fnd2,F12IND);
  a212=tex2D(localtex,fnd2,F21IND);
  a222=tex2D(localtex,fnd2,F22IND);
  xadd2=tex2D(localtex,fnd2,F13IND)+xcenin-xcenin*a112-slicen*a122;
  yadd2=tex2D(localtex,fnd2,F23IND)+slicen-xcenin*a212-slicen*a222;

  cbeta3=tex2D(localtex,fnd3,CBIND);
  sbeta3=tex2D(localtex,fnd3,SBIND);
  calf3=tex2D(localtex,fnd3,CAIND);
  salf3=tex2D(localtex,fnd3,SAIND);
  a113=tex2D(localtex,fnd3,F11IND);
  a123=tex2D(localtex,fnd3,F12IND);
  a213=tex2D(localtex,fnd3,F21IND);
  a223=tex2D(localtex,fnd3,F22IND);
  xadd3=tex2D(localtex,fnd3,F13IND)+xcenin-xcenin*a113-slicen*a123;
  yadd3=tex2D(localtex,fnd3,F23IND)+slicen-xcenin*a213-slicen*a223;

  cbeta4=tex2D(localtex,fnd4,CBIND);
  sbeta4=tex2D(localtex,fnd4,SBIND);
  calf4=tex2D(localtex,fnd4,CAIND);
  salf4=tex2D(localtex,fnd4,SAIND);
  a114=tex2D(localtex,fnd4,F11IND);
  a124=tex2D(localtex,fnd4,F12IND);
  a214=tex2D(localtex,fnd4,F21IND);
  a224=tex2D(localtex,fnd4,F22IND);
  xadd4=tex2D(localtex,fnd4,F13IND)+xcenin-xcenin*a114-slicen*a124;
  yadd4=tex2D(localtex,fnd4,F23IND)+slicen-xcenin*a214-slicen*a224;
       
  f1x=f1*a11;
  f2x=f2*a112;
  f3x=f3*a113;
  f4x=f4*a114;
  f1xy=f1*a12;
  f2xy=f2*a122;
  f3xy=f3*a123;
  f4xy=f4*a124;

  f1y=f1*a21;
  f2y=f2*a212;
  f3y=f3*a213;
  f4y=f4*a214;
  f1yy=f1*a22;
  f2yy=f2*a222;
  f3yy=f3*a223;
  f4yy=f4*a224;

  xalladd=f1*xadd+f2*xadd2+f3*xadd3+f4*xadd4;
  yalladd=f1*yadd+f2*yadd2+f3*yadd3+f4*yadd4;
       
  // Each projection position is a sum of a fixed factor ("..f")
  // and a factor that multiplies z ("..z")
   
  xx=j+1.-xcen;
  yy=lslice-slicen;
  xp1f=xx*cbeta + yy*salf*sbeta + xcenPdelxx;
  xp1z=calf*sbeta + tex2D(localtex,fnd1,XZFIND);
  xp2f=xx*cbeta2 + yy*salf2*sbeta2 + xcenPdelxx;
  xp2z=calf2*sbeta2 + tex2D(localtex,fnd2,XZFIND);
  xp3f=xx*cbeta3 + yy*salf3*sbeta3 + xcenPdelxx;
  xp3z=calf3*sbeta3 + tex2D(localtex,fnd3,XZFIND);
  xp4f=xx*cbeta4 + yy*salf4*sbeta4 + xcenPdelxx;
  xp4z=calf4*sbeta4 + tex2D(localtex,fnd4,XZFIND);

  yp1f=yy*calf + slicen;
  yp2f=yy*calf2 + slicen;
  yp3f=yy*calf3 + slicen;
  yp4f=yy*calf4 + slicen;

  // store the fixed and z-dependent component of the
  // projection coordinates
  yzf1 = tex2D(localtex,fnd1,YZFIND);
  yzf2 = tex2D(localtex,fnd2,YZFIND);
  yzf3 = tex2D(localtex,fnd3,YZFIND);
  yzf4 = tex2D(localtex,fnd4,YZFIND);
  ind1 = pitch * iv + j;
  xprjf[ind1] =f1x*xp1f+f2x*xp2f+f3x*xp3f+f4x*xp4f+
    f1xy*yp1f+f2xy*yp2f+f3xy*yp3f+f4xy*yp4f+xalladd;
  xprjz[ind1] =f1x*xp1z+f2x*xp2z+f3x*xp3z+f4x*xp4z-
    (f1xy*(salf-yzf1)+f2xy*(salf2-yzf2)+ f3xy*(salf3-yzf3)+f4xy*(salf4-yzf4));
  yprjf[ind1] =f1y*xp1f+f2y*xp2f+f3y*xp3f+f4y*xp4f+
    f1yy*yp1f+f2yy*yp2f+f3yy*yp3f+f4yy*yp4f+yalladd;
  yprjz[ind1] =f1y*xp1z+f2y*xp2z+f3y*xp3z+f4y*xp4z-
    (f1yy*(salf-yzf1)+f2yy*(salf2-yzf2)+ f3yy*(salf3-yzf3)+f4yy*(salf4-yzf4));
}

// Function to load the local alignment data
int gpuloadlocals(float *packed, int *numWarps)
{
  size_t sizetmp = sizeof(float) * *numWarps * numViews * 12;
  if (hipMemcpyToArray(localData, 0, 0, packed, sizetmp,
                        hipMemcpyHostToDevice) != hipSuccess) {
    pflerr("Failed to copy local data to GPU array");
    gpudone();
    return 1;
  }
  return 0;
}

// Function to run back-projection with local alignments, first computing the
// the projection factors for all positions and views, then running the 
// back projection kernel
int gpubplocal(float *slice, int *lslice, int *nxwarp, int *nywarp,
               int *ixswarp, int *iyswarp, int *idxwarp, int *idywarp,
               int *nxprj, float *xcen, float *xcenin, float *delxx,
               float *ycen, float *slicen, float *edgefill)
{
  int blockX = 16;

  // Compute the local projection factors
  dim3 blockFac(blockX, 16, 1);
  dim3 gridFac((sliceWidth + blockFac.x - 1) / blockFac.x, 
                (numViews + blockFac.y - 1) / blockFac.y, 1);
  localProjFactors<<<gridFac, blockFac>>>
    (xprojf, xprojz, yprojf, yprojz, slicePitch / 4, numViews, sliceWidth, 
     *lslice, *nxwarp, *nywarp, *ixswarp, *iyswarp, *idxwarp, *idywarp, *xcen,
     *xcenin, *xcenin+*delxx, *slicen);
  if (testReportErr("computing localProjFactors"))
      return 1;

  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after computing local factors");
    return 1;
  }


  // Do the backprojection
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (sliceThick + blockSize.y - 1) / blockSize.y, 1);

  bpLocalTest<<<gridSize, blockSize>>>
    (devSlice, xprojf, xprojz, yprojf, yprojz, slicePitch / 4, sliceWidth,
     *nxprj, lsliceFirst + numLoadedPlanes - 1, 
     sliceThick, numViews, *ycen, lsliceFirst, *edgefill);
  if (testReportErr("for local backprojection"))
      return 1;

  return (synchronizeCopySlice(slice, sliceThick));
}

/*
 * ROUTINES FOR REPROJECTION
 */

// Kernel to do simple reprojection (no X axis tilt or Z factors)
__global__ void reprojNox(float *lines, int pitch, int iwide, int ithick, 
                          int lsStart, int lsEnd, int lsliceBase, 
                          float xxlim, float xcenAdj, float xcenPdelxx,
                          float xprjOffset, float ycenAdj, float sbeta,
                          float cbetinv, float delz, int numz, float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, kz;
  float zz, sum, frac, zslice, xproj, xx;
  line = i + lsStart;
  sum = 0.;
  if (j >= iwide || line > lsEnd)
    return;
  for (kz = 0; kz < numz; kz++) {
    zz = 1 + kz * delz;
    frac = 1.;
    if (zz > ithick) {
      frac = 1. - (zz - (int)zz);
      zz = ithick;
    }
    zslice = zz - 0.5f;
    zz -= ycenAdj;
    xproj = j + 1 + xprjOffset;

    // the usual -0.5 is incorporated into xcenAdj
    xx = (xproj - (zz  * sbeta + xcenPdelxx)) * cbetinv + xcenAdj;
    if (xx < 0.5f || xx > xxlim) {
      sum += frac * pmean;
    } else {
      zslice += (line - lsliceBase) * ithick;
      sum += frac * tex2D(projtex, xx, zslice);
    }
  }
  lines[pitch * i + j] = sum;
}

// Kernel to do reprojection with X axis tilt and/or Z factors
__global__ void reprojXtilt(float *lines, int pitch, int iwide, int ithick, 
                            int lsStart, int lsEnd, int lsliceBase,
                            int lsliceLast, float xxlim, float xcenAdj,
                            float xcenPdelxx, float xprjOffset, float slicen,
                            float yprjOffset, float ycenAdj, float cbetinv,
                            float calfinv, float salfmyz, float salfsbet,
                            float calsbetpxz, float delz, int numz,
                            float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, kz,iys;
  float zz, sum, frac, zslice, yproj, yy, yslice, xproj, xx, fy;
  float ytol = 0.05f;
  line = i + lsStart;
  sum = 0.;
  if (j >= iwide || line > lsEnd)
    return;
  for (kz = 0; kz < numz; kz++) {
    zz = 1 + kz * delz;
    frac = 1.;
    if (zz > ithick) {
      frac = 1. - (zz - (int)zz);
      zz = ithick;
    }
    zslice = zz - 0.5f;
    zz -= ycenAdj;
    yproj = line + yprjOffset;
    yy = (yproj + zz * salfmyz - slicen) * calfinv;
    yslice = yy + slicen - yprjOffset;
    xproj = j + 1 + xprjOffset;

    // the usual -0.5 is incorporated into xcenAdj
    xx = (xproj - (yy * salfsbet + zz * calsbetpxz + xcenPdelxx)) * cbetinv +
      xcenAdj;
    if (xx < 0.5f || xx > xxlim || yslice < lsliceBase - ytol ||
        yslice > lsliceLast + ytol) {
      sum += frac * pmean;
    } else {
      iys = (int)yslice;
      if (iys < lsliceBase) {
        iys = lsliceBase;
        fy = 0.;
      } else if (iys >= lsliceLast) {
        iys = lsliceLast - 1;
        fy = 1.;
      } else {
        fy = yslice - iys;
      }
      zslice += (iys - lsliceBase) * ithick;
      sum += frac * ((1. - fy) * tex2D(projtex, xx, zslice) + 
                     fy * tex2D(projtex, xx, zslice + ithick));
    }
  }
  lines[pitch * i + j] = sum;
}

// Function to run reprojection for all cases except local alignments
int gpureproject(float *lines, float *sbeta, float *cbeta, float *salpha, 
                 float *calpha, float *xzfac, float *yzfac, float *delz,
                 int *lsStart, int *lsEnd, int *ithick,
                 float *xcen, float *xcenPdelxx, int *minXreproj, 
                 float *xprjOffset, float *ycen, int *minYreproj,
                 float *yprjOffset, float *slicen, int *ifalpha, float *pmean)
{ 
  int blockX = 16;
  int numz, numLines = *lsEnd + 1 - *lsStart;
  int lastSlice = lsliceFirst + numLoadedPlanes - 1;
  float znum, xcenAdj, salfsbet, calsbetpxz, ycenAdj, salfmyz, cbetinv,calfinv;
  znum = 1. + (*ithick - 1) / *delz;
  numz = znum;
  if (znum - numz > 0.1)
    numz++;
  xcenAdj = *xcen - (*minXreproj-1) - 0.5;
  salfsbet = *salpha * *sbeta;
  calsbetpxz = *calpha * *sbeta + *xzfac;
  ycenAdj = *ycen + 1 - *minYreproj;
  salfmyz = *salpha - *yzfac;
  cbetinv = 1. / *cbeta;
  calfinv = 1. / *calpha;

  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sliceWidth + blockSize.x - 1) / blockSize.x, 
                (numLines + blockSize.y - 1) / blockSize.y, 1);

  if (*ifalpha) {
    reprojXtilt<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, sliceWidth, *ithick, *lsStart, *lsEnd, 
       lsliceFirst, lastSlice, nxPlane - 0.5, xcenAdj, *xcenPdelxx,
       *xprjOffset, *slicen, *yprjOffset, ycenAdj, cbetinv, calfinv, salfmyz,
       salfsbet, calsbetpxz, *delz, numz, *pmean);
  } else {
    reprojNox<<<gridSize, blockSize>>>
      (devSlice, slicePitch / 4, sliceWidth, *ithick, *lsStart, *lsEnd, 
       lsliceFirst, nxPlane - 0.5, xcenAdj, *xcenPdelxx, *xprjOffset,
       ycenAdj, *sbeta, cbetinv, *delz, numz, *pmean);
  }
  if (testReportErr("for reprojection"))
    return 1;
  return (synchronizeCopySlice(lines, numLines));
}


/*
 * UTILITY ROUTINES
 */
   
// Load cosine and sine beta into constant array and compute inverse cosine
static int loadBetaInvertCos(float *cbeta, float *sbeta, float *cosinv)
{
  int i, iv;
  float yy;

  // Invert cosines with limit
  for (i = 0; i < numViews; i++) {
    yy = cbeta[i];
    if (fabs(yy) < 0.001f)
      yy = yy >= 0 ? 0.001f : -0.001f;
    cosinv[i] = 1.f / yy;
  }

  // Copy sines/cosines
  iv = numViews * sizeof(float);
  if (hipMemcpyToSymbol(HIP_SYMBOL(tables), cbeta, iv, 0, hipMemcpyHostToDevice) ||
      hipMemcpyToSymbol(HIP_SYMBOL(tables), sbeta, iv, SINOFS*4,
                            hipMemcpyHostToDevice)) {
    pflerr("Failed to copy constant data to GPU");
    return 1;
  }
  return 0;
}

// Synchronize the threads and copy computed data back to caller's array
static int synchronizeCopySlice(float *slice, int numLines)
{
  int sizetmp;
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after backprojection");
    return 1;
  }

  // Get slice back
  sizetmp = sizeof(float) * sliceWidth;
  if (hipMemcpy2D(slice, sizetmp, devSlice, slicePitch, sizetmp, numLines, 
                   hipMemcpyDeviceToHost) != hipSuccess) {
    pflerr("Error copying slice back to host");
    return 1;
  }
  return 0;
}

// Test for and report error after executing threads           
static int testReportErr(char *mess)
{
  hipError_t err;
  err = hipGetLastError();
  if (err != hipSuccess) {
    pflush("Error executing threads %s: %s\n", mess,
           hipGetErrorString(err));
    return 1;
  }
  return 0;
}

// Print a message with flushes to get it out before fortran output
static void pflush(const char *format, ...)
{
  char errorMess[512];
  va_list args;
  va_start(args, format);
  vsprintf(errorMess, format, args);
  printf(errorMess);
  fflush(stdout);  
  fflush(stdout);
  va_end(args);
}

// In case of error, find the error string and print it with message
static void pflerr(const char *format, ...)
{
  hipError_t err;
  char errorMess[512];
  va_list args;
  va_start(args, format);
  vsprintf(errorMess, format, args);
  printf(errorMess);
  err = hipGetLastError();
  pflush(": %s\n", hipGetErrorString(err));
  fflush(stdout);  
  fflush(stdout);
  va_end(args);
}

/*

$Log$

*/
